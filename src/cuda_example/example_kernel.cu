#include "hip/hip_runtime.h"
// if we don't include that Qt Creator will show many errors
#include <hip/hip_runtime.h>

static const int blocksize = 16;

__global__ void hello(char* a, int* b) {
  // increment chars with ints
  a[threadIdx.x] += b[threadIdx.x];
}

void run_kernel(char* str, int* val, int str_size, int val_size) {

  // CUDA side variables
  char* cuda_str;
  int* cuda_val;

  hipMalloc((void**)&cuda_str, str_size);
  hipMalloc((void**)&cuda_val, val_size);
  hipMemcpy(cuda_str, str, str_size, hipMemcpyHostToDevice);
  hipMemcpy(cuda_val, val, val_size, hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);
  // Qt Creator does not like that, but ask NVIDIA about fancy notation
  hello << <dimGrid, dimBlock>>> (cuda_str, cuda_val);

  hipMemcpy(str, cuda_str, str_size, hipMemcpyDeviceToHost);

  hipFree(cuda_str);
  hipFree(cuda_str);
}
