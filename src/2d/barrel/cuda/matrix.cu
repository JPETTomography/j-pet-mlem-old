#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "util/cuda/memory.h"
#include "util/random.h"

#include "matrix.h"

namespace PET2D {
namespace Barrel {
namespace GPU {

__global__ static void kernel(const Pixel pixel,
                              const DetectorRing* detector_ring_ptr,
                              int n_emissions,
                              float pixel_size,
                              int n_positions,
                              float tof_step,
                              float length_scale,
                              unsigned int* gpu_prng_seed,
                              int* pixel_hits) {

  bool tof = tof_step > 0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  util::random::tausworthe gen(&gpu_prng_seed[4 * tid]);
  util::random::uniform_real_distribution<float> one_dis(0, 1);

  __shared__ util::cuda::copy<DetectorRing> detector_ring_copier;
  detector_ring_copier = detector_ring_ptr;
  DetectorRing& detector_ring = *detector_ring_copier;

  Model model(length_scale);

  for (int i = 0; i < n_emissions; ++i) {
    auto rx = (pixel.x + one_dis(gen)) * pixel_size;
    auto ry = (pixel.y + one_dis(gen)) * pixel_size;
    auto angle = one_dis(gen) * (float)M_PI;

    // ensure we are within a triangle
    if (rx > ry)
      continue;

    LOR lor;
    float position = 0;
    Event event(rx, ry, angle);
    auto hits = detector_ring.detect(gen, model, event, lor, position);

    int quantized_position = 0;
    if (tof)
      quantized_position =
          detector_ring.quantize_position(position, tof_step, n_positions);

    // do we have hit on both sides?
    if (hits >= 2) {
      auto pixel_index =
          blockIdx.x * (lor.index() * n_positions + quantized_position);
      atomicAdd(&pixel_hits[pixel_index], 1);
    }
  }

  gen.save(&gpu_prng_seed[4 * tid]);
}

Matrix::Matrix(const DetectorRing& detector_ring,
               int n_threads_per_block,
               int n_blocks,
               float pixel_size,
               int n_positions,
               float tof_step,
               float length_scale,
               unsigned int* prng_seed)
    : n_threads_per_block(n_threads_per_block),
      n_blocks(n_blocks),
      pixel_size(pixel_size),
      n_positions(n_positions),
      tof_step(tof_step),
      length_scale(length_scale),
      pixel_hits_count(detector_ring.n_lors * n_positions),
      pixel_hits_size(pixel_hits_count * sizeof(int)),
      output_size(n_blocks * pixel_hits_size) {

  hipMalloc((void**)&gpu_detector_ring, sizeof(DetectorRing));
  hipMemcpy(gpu_detector_ring,
             &detector_ring,
             sizeof(DetectorRing),
             hipMemcpyHostToDevice);

  output = new int[n_blocks * detector_ring.n_lors * n_positions];
  hipMalloc((void**)&gpu_output, output_size);

  int prng_seed_size = n_blocks * n_threads_per_block * 4 * sizeof(*prng_seed);
  hipMalloc((void**)&gpu_prng_seed, prng_seed_size);
  hipMemcpy(gpu_prng_seed, prng_seed, prng_seed_size, hipMemcpyHostToDevice);
}

Matrix::~Matrix() {
  hipFree(gpu_prng_seed);
  hipFree(gpu_output);
  delete[] output;
  hipFree(gpu_detector_ring);
}

void Matrix::operator()(Pixel pixel, int n_emissions, int* pixel_hits) {

  hipMemset(gpu_output, 0, output_size);

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#define kernel kernel << <blocks, threads>>>
#endif
  kernel(pixel,
         gpu_detector_ring,
         n_emissions,
         pixel_size,
         n_positions,
         tof_step,
         length_scale,
         gpu_prng_seed,
         gpu_output);

  hipDeviceSynchronize();
  hipMemcpy(output, gpu_output, output_size, hipMemcpyDeviceToHost);

  // Reduce blocks into pixel hits:
  // 1. First block can be simply copied
  memcpy(pixel_hits, output, pixel_hits_size);
  // 2. Next blocks must be reduced
  for (int block = 1; block < n_blocks; ++block) {
    for (int i = 0; i < pixel_hits_count; ++i) {
      pixel_hits[i] += output[block * pixel_hits_count + i];
    }
  }
}

}  // GPU
}  // Barrel
}  // PET2D
