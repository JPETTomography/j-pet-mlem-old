#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "util/cuda/memory.h"
#include "util/random.h"

#include "matrix.h"

namespace PET2D {
namespace Barrel {
namespace GPU {
namespace Matrix {

__global__ static void kernel(const Pixel pixel,
                              const Scanner* scanner_ptr,
                              int n_emissions,
                              float s_pixel,
                              int n_tof_positions,
                              float tof_step,
                              float length_scale,
                              unsigned int* gpu_rng_seed,
                              int* pixel_hits) {
  bool tof = tof_step > 0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  util::random::tausworthe rng(&gpu_rng_seed[4 * tid]);
  util::random::uniform_real_distribution<float> one_dis(0, 1);
  util::random::uniform_real_distribution<float> pi_dis(0, (float)M_PI);

  __shared__ util::cuda::copy<Scanner> scanner_shared_storage;
  scanner_shared_storage = scanner_ptr;
  Scanner& scanner = *scanner_shared_storage;

  Model model(length_scale);
  auto fov_radius2 = scanner.fov_radius() * scanner.fov_radius();

  for (int i = 0; i < n_emissions; ++i) {
    auto rx = (pixel.x + one_dis(rng)) * s_pixel;
    auto ry = (pixel.y + one_dis(rng)) * s_pixel;
    auto angle = pi_dis(rng);

    // ensure we are within a triangle
    if (rx > ry)
      continue;

    // ensure we are within FOV
    if (rx * rx + ry * ry > fov_radius2)
      continue;

    Event event(rx, ry, angle);
    Scanner::Response response;
    auto hits = scanner.detect(rng, model, event, response);

    int quantized_position = 0;
    if (tof)
      quantized_position = Scanner::quantize_tof_position(
          response.dl, tof_step, n_tof_positions);

    // do we have hit on both sides?
    if (hits >= 2) {
      auto pixel_index =
          response.lor.index() * n_tof_positions + quantized_position;
      atomicAdd(&pixel_hits[pixel_index], 1);
    }
  }

  rng.save(&gpu_rng_seed[4 * tid]);
}

template <>
void run<Scanner>(Scanner& scanner,
                  util::random::tausworthe& rng,
                  int n_blocks,
                  int n_threads_per_block,
                  int n_emissions,
                  double tof_step,
                  int n_tof_positions,
                  int n_pixels,
                  double s_pixel,
                  double length_scale,
                  util::delegate<void(int, bool)> progress,
                  util::delegate<void(LOR, S, Pixel, Hit)> entry) {

  // GTX 770 - 8 SMX * 192 cores = 1536 cores -
  // each SMX can use 8 active blocks,
  auto n_threads = n_blocks * n_threads_per_block;
  auto n_thread_emissions = (n_emissions + n_threads - 1) / n_threads;
  // Number of emissions will be rounded to block size
  n_emissions = n_thread_emissions * n_threads;

  // (1) setup LORs:
  const auto end_lor = LOR::end_for_detectors(scanner.size());
  const auto n_lors = end_lor.index();
  LOR lor_map[n_lors];
  for (LOR lor(0, 0); lor < end_lor; ++lor) {
    lor_map[lor.index()] = lor;
  }

  // (2) copy scanner to GPU:
  Scanner* gpu_scanner;
  hipMalloc((void**)&gpu_scanner, sizeof(Scanner));
  hipMemcpy(gpu_scanner, &scanner, sizeof(Scanner), hipMemcpyHostToDevice);

  // (3) create empty pixel hits for GPU:
  const int pixel_hits_len = n_lors * n_tof_positions;
  const int pixel_hits_size = pixel_hits_len * sizeof(int);
  int* gpu_pixel_hits;
  int pixel_hits[pixel_hits_len];
  hipMalloc((void**)&gpu_pixel_hits, pixel_hits_size);

  // (4) initalize RNG for all GPU threads:
  int rng_seed_len = n_threads * 4;
  unsigned int rng_seed[rng_seed_len];
  int rng_seed_size = rng_seed_len * sizeof(*rng_seed);
  for (int i = 0; i < rng_seed_len; ++i) {
    rng_seed[i] = rng();
  }
  unsigned int* gpu_rng_seed;
  hipMalloc((void**)&gpu_rng_seed, rng_seed_size);
  hipMemcpy(gpu_rng_seed, rng_seed, rng_seed_size, hipMemcpyHostToDevice);

  auto end_pixel = Pixel::end_for_n_pixels_in_row(n_pixels / 2);
  for (Pixel pixel(0, 0); pixel < end_pixel; ++pixel) {
    progress(pixel.index(), false);

    // (5) clear pixel hits on GPU:
    hipMemset(gpu_pixel_hits, 0, pixel_hits_size);

#if __HIPCC__
    dim3 blocks(n_blocks);
    dim3 threads(n_threads_per_block);
#define kernel kernel<<<blocks, threads>>>
#endif
    // (6) run kernel:
    kernel(pixel,
           gpu_scanner,
           n_emissions,
           s_pixel,
           n_tof_positions,
           tof_step,
           length_scale,
           gpu_rng_seed,
           gpu_pixel_hits);

    // (7) copy back the data:
    hipDeviceSynchronize();
    hipMemcpy(
        pixel_hits, gpu_pixel_hits, pixel_hits_size, hipMemcpyDeviceToHost);

    // (8) inform callee about data:
    for (size_t lor_index = 0; lor_index < n_lors; ++lor_index) {
      auto lor = lor_map[lor_index];
      for (int position = 0; position < n_tof_positions; ++position) {
        auto hits = pixel_hits[n_tof_positions * lor_index + position];
        if (hits > 0) {
          entry(lor, position, pixel, hits);
        }
      }
    }

    progress(pixel.index(), true);
  }

  // (9) free the GPU data:
  hipFree(gpu_rng_seed);
  hipFree(gpu_pixel_hits);
  hipFree(gpu_scanner);
}

}  // Matrix
}  // GPU
}  // Barrel
}  // PET2D
