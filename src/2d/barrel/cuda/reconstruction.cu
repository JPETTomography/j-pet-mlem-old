#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "util/cuda/memory.h"
#include "util/delegate.h"

#include "common/cuda/kernels.h"

#include "reconstruction.h"

namespace PET2D {
namespace Barrel {
namespace GPU {
namespace Reconstruction {

texture<F, 2, hipReadModeElementType> tex_rho;

// foreach p: count y[p] and store it in output_rho[p]
__global__ static void reconstruction_1(const PixelInfo* pixel_infos,
                                        const size_t* lor_pixel_info_start,
                                        const size_t* lor_pixel_info_end,
                                        const Mean* means,
                                        const int n_means,
                                        F* output_rho,
                                        const int width) {

  const auto tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto n_threads = gridDim.x * blockDim.x;
  const auto n_chunks = (n_means + n_threads - 1) / n_threads;

  for (int chunk = 0; chunk < n_chunks; ++chunk) {
    int mean_index = chunk * n_threads + tid;
    // check if we are still on the list
    if (mean_index >= n_means) {
      break;
    }

    auto mean = means[mean_index];
    auto lor_index = mean.lor.index();
    auto pixel_info_start = lor_pixel_info_start[lor_index];
    auto pixel_info_end = lor_pixel_info_end[lor_index];

    // count u for current lor
    F u = 0;
    for (auto i = pixel_info_start; i < pixel_info_end; ++i) {
      auto pixel_info = pixel_infos[i];
      auto pixel = pixel_info.pixel;
      u += tex2D(tex_rho, pixel.x, pixel.y) * pixel_info.weight;
    }
    F phi = mean.mean / u;
    for (auto i = pixel_info_start; i < pixel_info_end; ++i) {
      auto pixel_info = pixel_infos[i];
      auto pixel = pixel_info.pixel;
      atomicAdd(&output_rho[pixel.y * width + pixel.x],
                phi * pixel_info.weight);
    }
  }
}

// foreach p: count output_rho[p] *= rho[p]
__global__ static void reconstruction_2(F* output_rho,
                                        const F* scale,
                                        const int width,
                                        const int height) {

  const auto tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto n_threads = gridDim.x * blockDim.x;
  const auto n_pixels = width * height;
  const auto n_pixel_chunks = (n_pixels + n_threads - 1) / n_threads;

  for (int chunk = 0; chunk < n_pixel_chunks; ++chunk) {
    int pixel_index = chunk * n_threads + tid;
    // check if we are still on the list
    if (pixel_index >= n_pixels) {
      break;
    }
    Pixel pixel(pixel_index % width, pixel_index / width);
    // there is no collision there, so we don't need atomics
    output_rho[pixel_index] *=
        tex2D(tex_rho, pixel.x, pixel.y) * scale[pixel_index];
  }
}

void run(const SimpleGeometry& geometry,
         const Mean* means,
         int n_means,
         int width,
         int height,
         int n_iteration_blocks,
         int n_iterations_in_block,
         util::delegate<void(int iteration, const Output& output)> output,
         util::delegate<void(int completed, bool finished)> progress,
         int device,
         int n_blocks,
         int n_threads_per_block,
         util::delegate<void(const char* device_name)> info) {

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#define sensitivity sensitivity<<<blocks, threads>>>
#define invert invert<<<blocks, threads>>>
#define reconstruction_1 reconstruction_1<<<blocks, threads>>>
#define reconstruction_2 reconstruction_2<<<blocks, threads>>>
#else
  (void)n_blocks, n_threads_per_block;  // mark used
#endif

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  info(prop.name);

  util::cuda::on_device<PixelInfo> device_pixel_infos(geometry.pixel_infos,
                                                      geometry.n_pixel_infos);
  util::cuda::on_device<size_t> device_lor_pixel_info_start(
      geometry.lor_pixel_info_start, geometry.n_lors);
  util::cuda::on_device<size_t> device_lor_pixel_info_end(
      geometry.lor_pixel_info_end, geometry.n_lors);
  util::cuda::on_device<Mean> device_means(means, n_means);

  util::cuda::memory2D<F> rho(tex_rho, width, height);
  util::cuda::on_device<F> output_rho((size_t)width * height);

  for (auto& v : rho) {
    v = 1;
  }
  rho.copy_to_device();

  util::cuda::on_device<F> scale((size_t)width * height);
  scale.zero_on_device();

  Common::GPU::sensitivity(
      device_pixel_infos, geometry.n_pixel_infos, scale, width);
  hipDeviceSynchronize();

  Common::GPU::invert(scale, width * height);
  hipDeviceSynchronize();

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {
      progress(ib * n_iterations_in_block + it, false);

      output_rho.zero_on_device();

      reconstruction_1(device_pixel_infos,
                       device_lor_pixel_info_start,
                       device_lor_pixel_info_end,
                       device_means,
                       n_means,
                       output_rho,
                       width);
      hipDeviceSynchronize();

      reconstruction_2(output_rho, scale, width, height);
      hipDeviceSynchronize();

      rho = output_rho;
      progress(ib * n_iterations_in_block + it, true);
    }

    rho.copy_from_device();
    Output rho_output(width, height, rho.host_ptr);
    output((ib + 1) * n_iterations_in_block, rho_output);
  }

  progress(n_iteration_blocks * n_iterations_in_block, false);
}

}  // Reconstruction
}  // GPU
}  // Barrel
}  // PET2D
