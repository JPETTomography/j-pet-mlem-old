#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "../event.h"
#include "../kernel.h"
#include "gpu_events_soa.h"

texture<float, 2, hipReadModeElementType> tex_sensitivity;
texture<float, 2, hipReadModeElementType> tex_rho;

#if USE_WARP_GRANULARITY
#include "reconstruction_warp_granularity.cuh"
#elif USE_THREAD_GRANULARITY
#include "reconstruction_thread_granularity.cuh"
#else
#include "reconstruction_simple.cuh"
#endif

namespace PET2D {
namespace Strip {
namespace GPU {

template <typename F>
void fill_with_sensitivity(F* sensitivity, Detector<F,short>& detector);

template <typename F>
void run_reconstruction(Detector<F, short>& detector,
                        Event<F>* events,
                        int n_events,
                        int n_iteration_blocks,
                        int n_iterations_in_block,
                        void (*output_callback)(Detector<F, short>& detector,
                                                int iteration,
                                                F* image,
                                                void* context),
                        void (*progress_callback)(int iteration,
                                                  void* context,
                                                  bool finished),
                        void* context,
                        int device,
                        int n_blocks,
                        int n_threads_per_block,
                        bool verbose) {

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  if (verbose) {
    fprintf(stdout, "# running on: %s\n", prop.name);
  }

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#endif

  size_t image_size = detector.total_n_pixels * sizeof(F);

  const int width = detector.n_z_pixels;
  const int height = detector.n_y_pixels;

  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

  F* cpu_sensitivity = new F[detector.total_n_pixels];

  fill_with_sensitivity(cpu_sensitivity, detector);

  output_callback(detector, -1, cpu_sensitivity, context);

  F* gpu_sensitivity;
  size_t pitch_sensitivity;
  hipMallocPitch(
      &gpu_sensitivity, &pitch_sensitivity, sizeof(F) * width, height);
  hipMemcpy2D(gpu_sensitivity,
               pitch_sensitivity,
               cpu_sensitivity,
               sizeof(F) * width,
               sizeof(F) * width,
               height,
               hipMemcpyHostToDevice);
  delete[] cpu_sensitivity;

  hipBindTexture2D(NULL,
                    &tex_sensitivity,
                    gpu_sensitivity,
                    &desc,
                    width,
                    height,
                    pitch_sensitivity);

  F* cpu_rho = new F[detector.total_n_pixels];
  for (int i = 0; i < detector.total_n_pixels; ++i) {
    cpu_rho[i] = 100;
  }

  // this class allocated CUDA pointers and deallocated them in destructor
  GPU::EventsSOA<F> gpu_events(events, n_events);

  F* gpu_rho;
  size_t pitch_rho;
  hipMallocPitch(&gpu_rho, &pitch_rho, sizeof(F) * width, height);
  hipBindTexture2D(NULL, &tex_rho, gpu_rho, &desc, width, height, pitch_rho);

  F* gpu_output_rho;

#if USE_RHO_PER_WARP
  hipMalloc((void**)&gpu_output_rho, n_blocks * image_size);
  F* cpu_output_rho;
  cpu_output_rho = new F[n_blocks * detector.total_n_pixels];
#else
  hipMalloc((void**)&gpu_output_rho, image_size);
#endif

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {
      progress_callback(ib * n_iterations_in_block + it, context, false);

#if USE_RHO_PER_WARP
      hipMemset(gpu_output_rho, 0, n_blocks * image_size);
#else
      hipMemset(gpu_output_rho, 0, image_size);
#endif
      hipMemcpy2D(gpu_rho,
                   pitch_rho,
                   cpu_rho,
                   sizeof(F) * width,
                   sizeof(F) * width,
                   height,
                   hipMemcpyHostToDevice);

#if __HIPCC__
#define reconstruction reconstruction<Kernel> << <blocks, threads>>>
#endif
      reconstruction(detector,
                     gpu_events.z_u,
                     gpu_events.z_d,
                     gpu_events.dl,
                     n_events,
                     gpu_output_rho,
                     n_blocks,
                     n_threads_per_block);

      hipDeviceSynchronize();

#if USE_RHO_PER_WARP
      hipMemcpy(cpu_output_rho,
                 gpu_output_rho,
                 n_blocks * image_size,
                 hipMemcpyDeviceToHost);

      for (int i = 0; i < detector.n_y_pixels; ++i) {
        for (int j = 0; j < detector.n_z_pixels; ++j) {
          int pixel_adr = i * detector.n_y_pixels + j;
          cpu_rho[pixel_adr] = 0;
          for (int block_id = 0; block_id < n_blocks; ++block_id) {

            cpu_rho[i * detector.n_y_pixels + j] +=
                cpu_output_rho[block_id * detector.n_y_pixels + pixel_adr];
          }
        }
      }

#else
      hipMemcpy(cpu_rho, gpu_output_rho, image_size, hipMemcpyDeviceToHost);
#endif
      progress_callback(ib * n_iterations_in_block + it, context, true);
    }

    output_callback(detector, ib * n_iterations_in_block, cpu_rho, context);
  }

  progress_callback(n_iteration_blocks * n_iterations_in_block, context, false);

  hipUnbindTexture(&tex_sensitivity);
  hipFree(gpu_sensitivity);
  hipUnbindTexture(&tex_rho);
  hipFree(gpu_rho);
  hipFree(gpu_output_rho);
  delete[] cpu_rho;
#if USE_RHO_PER_WARP
  delete[] cpu_output_rho;
#endif
}

template <typename F>
void fill_with_sensitivity(F* sensitivity, Detector<F, short>& detector) {

  size_t width = detector.n_z_pixels;
  size_t height = detector.n_y_pixels;

  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      sensitivity[y * width + x] = detector.pixel_sensitivity(Pixel<short>(x, y));
    }
  }
}

// Explicit template instantiation

template void run_reconstruction<float>(
    Detector<float, short>& detector,
    Event<float>* events,
    int n_events,
    int n_iteration_blocks,
    int n_iterations_in_block,
    void (*output_callback)(Detector<float, short>& detector,
                            int iteration,
                            float* image,
                            void* context),
    void (*progress_callback)(int iteration, void* context, bool finished),
    void* context,
    int device,
    int n_blocks,
    int n_threads_per_block,
    bool verbose);

}  // GPU
}  // Strip
}  // PET2D
