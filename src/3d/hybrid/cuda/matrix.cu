#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "util/cuda/memory.h"
#include "util/random.h"

#include "matrix.h"

#include "3d/geometry/distribution.h"

namespace PET3D {
namespace Hybrid {
namespace GPU {
namespace Matrix {

__global__ static void kernel(const float z,
                              const Pixel pixel,
                              const Scanner* scanner_ptr,
                              int n_thread_emissions,
                              float s_pixel,
                              float length_scale,
                              util::random::tausworthe::state_type* rng_state,
                              int* pixel_hits) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  util::random::tausworthe rng(rng_state[tid]);
  util::random::uniform_real_distribution<float> one_dis(0, 1);
  util::random::uniform_real_distribution<float> pi_dis(0, (float)M_PI);
  Distribution::SphericalDistribution<F> direction;

  __shared__ util::cuda::copy<Scanner> scanner_shared_storage;
  scanner_shared_storage = scanner_ptr;
  Scanner& scanner = *scanner_shared_storage;

  Model model(length_scale);

  for (int i = 0; i < n_thread_emissions; ++i) {
    auto rx = (pixel.x + one_dis(rng)) * s_pixel;
    auto ry = (pixel.y + one_dis(rng)) * s_pixel;

    // ensure we are within a triangle, so we got only half hits on diagonal
    if (rx > ry)
      continue;

    auto rz = z + one_dis(rng) * s_pixel;

    Event event(PET3D::Point<float>(rx, ry, rz), direction(rng));
    Scanner::Response response;
    auto hits = scanner.detect(rng, model, event, response);

    // do we have hit on both sides?
    if (hits >= 2) {
      auto pixel_index = response.lor.index();
      atomicAdd(&pixel_hits[pixel_index], 1);
    }
  }

  rng.save(rng_state[tid]);
}

template <>
void run<Scanner>(
    Scanner& scanner,
    util::random::tausworthe& rng,
    int n_emissions,
    double z_position,
    int n_pixels,
    double s_pixel,
    double length_scale,
    util::delegate<void(int completed, bool finished)> progress,
    util::delegate<void(LOR lor, Pixel pixel, Hit hits)> entry,
    int device,
    int n_blocks,
    int n_threads_per_block,
    util::delegate<void(const char* device_name, int n_emissions)> info) {

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#define kernel kernel<<<blocks, threads>>>
#endif

  // GTX 770 - 8 SMX * 192 cores = 1536 cores -
  // each SMX can use 8 active blocks,
  auto n_threads = n_blocks * n_threads_per_block;
  auto n_thread_emissions = (n_emissions + n_threads - 1) / n_threads;
  // Number of emissions will be rounded to block size
  n_emissions = n_thread_emissions * n_threads;

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  info(prop.name, n_emissions);

  const auto end_lor = LOR::end_for_detectors(scanner.barrel.size());
  const auto n_lors = end_lor.index();
  LOR lor_map[n_lors];
  for (LOR lor(0, 0); lor < end_lor; ++lor) {
    lor_map[lor.index()] = lor;
  }

  util::cuda::on_device<Scanner> scanner_on_device(scanner);
  util::cuda::memory<int> pixel_hits(n_lors);
  util::cuda::memory<util::random::tausworthe::state_type> rng_state(n_threads);

  for (size_t i = 0; i < rng_state.size; ++i) {
    util::random::tausworthe thread_rng(rng);
    thread_rng.save(rng_state[i]);
  }
  rng_state.copy_to_device();

  const auto pixel_fov_radius = scanner.barrel.fov_radius() / s_pixel;
  const int pixel_fov_radius2 = pixel_fov_radius * pixel_fov_radius;

  auto end_pixel = Pixel::end_for_n_pixels_in_row(n_pixels / 2);
  for (Pixel pixel(0, 0); pixel < end_pixel; ++pixel) {
    // ensure we are within FOV
    if (pixel.distance_from_origin2() > pixel_fov_radius2)
      continue;

    progress(pixel.index(), false);

    pixel_hits.zero_on_device();

    kernel(z_position,
           pixel,
           scanner_on_device,
           n_thread_emissions,
           s_pixel,
           length_scale,
           rng_state,
           pixel_hits);
    hipPeekAtLastError();  // ensure kernel was run successfully
    hipDeviceSynchronize();

    pixel_hits.copy_from_device();

    for (int lor_index = 0; lor_index < n_lors; ++lor_index) {
      auto lor = lor_map[lor_index];
      auto hits = pixel_hits[lor_index];
      if (hits > 0) {
        entry(lor, pixel, hits);
      }
    }

    progress(pixel.index(), true);
  }
}

}  // Matrix
}  // GPU
}  // Hybrid
}  // PET3D
