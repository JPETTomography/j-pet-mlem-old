#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "util/cuda/memory.h"
#include "util/delegate.h"

#include "reconstruction.h"

#define USE_TEXTURE 1  // using textures is faster when using bigger rhos

#if USE_WARP_GRANULARITY
#if USE_VOXEL_GRANULARITY  // voxel (densier) granularity
#include "reconstruction/warp_voxel_granularity.cuh"
#else            // pixel granularity (faster)
#if USE_TEXTURE  // use texture and 3D arrays for rho and 2D sensitivity lookup
#include "reconstruction/warp_granularity.cuh"
#else  // use linear memory
#include "reconstruction/warp_granularity_no_tex.cuh"
#endif
#endif
#elif USE_THREAD_GRANULARITY
#include "reconstruction/thread_granularity.cuh"
#endif

#include "common/cuda/kernels.h"

namespace PET3D {
namespace Hybrid {
namespace GPU {
namespace Reconstruction {

__global__ static void add_offsets(Event* events,
                                   const int n_events,
                                   const size_t* lor_pixel_info_begin) {
  const auto tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto n_threads = gridDim.x * blockDim.x;
  const auto n_chunks = (n_events + n_threads - 1) / n_threads;
  for (int chunk = 0; chunk < n_chunks; ++chunk) {
    int event_index = chunk * n_threads + tid;
    // check if we are still on the list
    if (event_index >= n_events) {
      break;
    }
    auto& event = events[event_index];
    const auto pixel_info_begin = lor_pixel_info_begin[event.lor.index()];
    event.pixel_info_begin += pixel_info_begin;
    event.pixel_info_end += pixel_info_begin;
  }
}

void run(const SimpleGeometry& geometry,
         const Sensitivity& sensitivity,
         const Event* events,
         int n_events,
         F sigma_z,
         F sigma_dl,
         const Grid& grid,
         const F barrel_length,
         int n_iteration_blocks,
         int n_iterations_in_block,
         util::delegate<void(int iteration, const Output& output)> output,
         util::delegate<void(int completed, bool finished)> progress,
         int device,
         int n_blocks,
         int n_threads_per_block,
         util::delegate<void(const char* device_name)> info) {

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#define reduce_to_sensitivity reduce_to_sensitivity<<<blocks, threads>>>
#define invert invert<<<blocks, threads>>>
#define reconstruction reconstruction<<<blocks, threads>>>
#define add_offsets add_offsets<<<blocks, threads>>>
#else
  (void)n_blocks, n_threads_per_block;  // mark used
#endif

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  info(prop.name);

  util::cuda::on_device<LineSegment> device_lor_line_segments(
      geometry.lor_line_segments, geometry.n_lors);
  util::cuda::on_device<PixelInfo> device_pixel_infos(geometry.pixel_infos,
                                                      geometry.n_pixel_infos);
  util::cuda::on_device<size_t> device_lor_pixel_info_begin(
      geometry.lor_pixel_info_begin, geometry.n_lors);
  util::cuda::on_device<Event> device_events(events, n_events);

  add_offsets(device_events, n_events, device_lor_pixel_info_begin);

#if USE_TEXTURE
  util::cuda::texture3D<F> rho(tex_rho,
                               grid.pixel_grid.n_columns,
                               grid.pixel_grid.n_rows,
                               grid.n_planes);
#else
  util::cuda::on_device<F> rho((size_t)grid.n_voxels);
#endif
  util::cuda::memory<F> output_rho((size_t)grid.n_voxels);
  Output rho_output(grid.pixel_grid.n_columns,
                    grid.pixel_grid.n_rows,
                    grid.n_planes,
                    output_rho.host_ptr);

  for (auto& v : output_rho) {
    v = 1;
  }
  output_rho.copy_to_device();

#if USE_TEXTURE
  util::cuda::texture2D<F> device_sensitivity(tex_sensitivity,
                                              (size_t)sensitivity.width,
                                              (size_t)sensitivity.height,
                                              sensitivity.data);
#else
  util::cuda::on_device<F> device_sensitivity((size_t)grid.pixel_grid.n_pixels);
#endif
  (void)device_sensitivity;  // device sensitivity is used via tex_sensitivity

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {
      progress(ib * n_iterations_in_block + it, false);

      rho = output_rho;
      output_rho.zero_on_device();

      reconstruction(device_lor_line_segments,
                     device_pixel_infos,
                     device_events,
                     n_events,
                     output_rho,
#if !USE_TEXTURE
                     rho,
                     device_sensitivity,
#endif
                     sigma_z,
                     sigma_dl,
                     grid,
                     barrel_length);
      hipDeviceSynchronize();

      progress(ib * n_iterations_in_block + it, true);

      // always output first 5 iterations, and at 10, 15, 20, 30, 50, 100
      if (!ib && it < n_iterations_in_block - 1 &&
          (it < 5 || it == 9 || it == 14 || it == 19 || it == 29 || it == 49 ||
           it == 99)) {
        output_rho.copy_from_device();
        output(it + 1, rho_output);
      }
    }

    output_rho.copy_from_device();
    output((ib + 1) * n_iterations_in_block, rho_output);
  }

  progress(n_iteration_blocks * n_iterations_in_block, false);
}

}  // Reconstruction
}  // GPU
}  // Hybrid
}  // PET3D
