#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "util/cuda/memory.h"
#include "util/delegate.h"

#include "common/cuda/kernels.h"

#include "reconstruction.h"

namespace PET3D {
namespace Hybrid {
namespace GPU {
namespace Reconstruction {

texture<float, 3, hipReadModeElementType> tex_rho;

__global__ static void reconstruction(const PixelInfo* pixel_infos,
                                      const Event* events,
                                      const int n_events,
                                      float* output_rho,
                                      const float* scale,
                                      const float sigma_z,
                                      const float sigma_dl,
                                      const int width,
                                      const int height) {

  const auto tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto n_threads = gridDim.x * blockDim.x;
  const auto n_chunks = (n_events + n_threads - 1) / n_threads;

  // --- event loop ----------------------------------------------------------
  for (int chunk = 0; chunk < n_chunks; ++chunk) {
    int event_index = chunk * n_threads + tid;
    // check if we are still on the list
    if (event_index >= n_events) {
      break;
    }

    const auto event = events[event_index];
    (void)event;  // FIXME: implement me!

  }  // event loop
}

void run(const SimpleGeometry& geometry,
         const Event* events,
         int n_events,
         float sigma_z,
         float sigma_dl,
         int width,
         int height,
         int depth,
         int n_iteration_blocks,
         int n_iterations_in_block,
         util::delegate<void(int iteration, const Output& output)> output,
         util::delegate<void(int completed, bool finished)> progress,
         int device,
         int n_blocks,
         int n_threads_per_block,
         util::delegate<void(const char* device_name)> info) {

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#define sensitivity sensitivity<<<blocks, threads>>>
#define invert invert<<<blocks, threads>>>
#define reconstruction reconstruction<<<blocks, threads>>>
#define add_offsets add_offsets<<<blocks, threads>>>
#else
  (void)n_blocks, n_threads_per_block;  // mark used
#endif

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  info(prop.name);

  util::cuda::on_device<PixelInfo> device_pixel_infos(geometry.pixel_infos,
                                                      geometry.n_pixel_infos);
  util::cuda::on_device<size_t> device_lor_pixel_info_start(
      geometry.lor_pixel_info_start, geometry.n_lors);
  util::cuda::on_device<Event> device_events(events, n_events);

  util::cuda::memory3D<F> rho(tex_rho, width, height, depth);
  util::cuda::on_device<F> output_rho((size_t)width * height * depth);

  for (auto& v : rho) {
    v = 1;
  }
  rho.copy_to_device();

  util::cuda::on_device<F> scale((size_t)width * height);
  scale.zero_on_device();

  Common::GPU::sensitivity(
      device_pixel_infos, geometry.n_pixel_infos, scale, width);
  hipDeviceSynchronize();

  Common::GPU::invert(scale, width * height);
  hipDeviceSynchronize();

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {
      progress(ib * n_iterations_in_block + it, false);

      output_rho.zero_on_device();

      reconstruction(device_pixel_infos,
                     device_events,
                     n_events,
                     output_rho,
                     scale,
                     sigma_z,
                     sigma_dl,
                     width,
                     height);
      hipDeviceSynchronize();

      rho = output_rho;
      progress(ib * n_iterations_in_block + it, true);
    }

    rho.copy_from_device();
    Output rho_output(width, height, depth, rho.host_ptr);
    output((ib + 1) * n_iterations_in_block, rho_output);
  }

  progress(n_iteration_blocks * n_iterations_in_block, false);
}

}  // Reconstruction
}  // GPU
}  // Hybrid
}  // PET3D
