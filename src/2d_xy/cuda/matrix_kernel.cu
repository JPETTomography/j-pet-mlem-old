#include <hip/hip_runtime.h>
#include <stdio.h>

#include "config.h"
#include "prng.cuh"
#include "geometry.h"
#include "geometry_methods.cuh"
#include "monte_carlo.cuh"

#include "geometry/pixel.h"

using namespace gpu;

static hipError_t err;

#define cuda(f, ...)                                        \
  if ((err = cuda##f(__VA_ARGS__)) != hipSuccess) {        \
    fprintf(stderr, #f "() %s\n", hipGetErrorString(err)); \
    exit(-1);                                               \
  }
#define cudathread_per_blockoSync(...) cuda(__VA_ARGS__)

bool run_monte_carlo_kernel(int pixel_i,
                            int n_tof_positions,
                            int number_of_threads_per_block,
                            int number_of_blocks,
                            int n_emissions,
                            float radius,
                            float h_detector,
                            float w_detector,
                            float pixel_size,
                            gpu::LOR* lookup_table_lors,
                            Pixel<>* lookup_table_pixel,
                            unsigned int* cpu_prng_seed,
                            MatrixElement* cpu_matrix,
                            MatrixElement* gpu_output) {

  dim3 blocks(number_of_blocks);
  dim3 threads(number_of_threads_per_block);

  hipSetDevice(0);

  unsigned int* gpu_prng_seed;
  MatrixElement* gpu_MatrixElement;

#if WARP_DIVERGENCE_TEST
  bool* warp_divergence_buffor;

  const int warp_size = 32;

  cuda(Malloc,
       (void**)&warp_divergence_buffor,
       warp_size * n_emissions * sizeof(bool));

  cuda(Memset,
       warp_divergence_buffor,
       0,
       warp_size * n_emissions * sizeof(bool));

#else
  bool* warp_divergence_buffor;
  cuda(Malloc, (void**)&warp_divergence_buffor, 0 * sizeof(bool));

#endif

  cuda(Malloc,
       (void**)&gpu_prng_seed,
       number_of_blocks * number_of_threads_per_block * 4 *
           sizeof(unsigned int));
  cuda(Malloc,
       (void**)&gpu_MatrixElement,
       n_tof_positions * number_of_blocks * sizeof(MatrixElement));

  cuda(
      Memcpy,
      gpu_prng_seed,
      cpu_prng_seed,
      number_of_blocks * number_of_threads_per_block * 4 * sizeof(unsigned int),
      hipMemcpyHostToDevice);

  float fov_radius = radius / M_SQRT2;

  Pixel<> pixel = lookup_table_pixel[pixel_i];

  int i = pixel.x;
  int j = pixel.y;

  cuda(Memset,
       gpu_MatrixElement,
       0,
       n_tof_positions * number_of_blocks * sizeof(MatrixElement));

  long total_emissions =
      (long)n_emissions * number_of_blocks * number_of_threads_per_block;

  printf(
      "Pixel(%d,%d) n_emissions: %d %ld\n", i, j, n_emissions, total_emissions);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  if ((i * i + j * j) * pixel_size * pixel_size < fov_radius * fov_radius) {

    monte_carlo_kernel << <blocks, threads>>> (i,
                                               j,
                                               n_emissions,
                                               n_tof_positions,
                                               gpu_prng_seed,
                                               gpu_MatrixElement,
                                               radius,
                                               h_detector,
                                               w_detector,
                                               pixel_size,
                                               warp_divergence_buffor);

    hipDeviceSynchronize();

    if (hipGetLastError() != hipSuccess) {
      return false;
    }
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Direct kernel time without memcpy %f ms\n", milliseconds);

#if WARP_DIVERGENCE_TEST

  bool* cpu_warp_divergence_buffor = new bool[warp_size * n_emissions];

  cuda(Memcpy,
       cpu_warp_divergence_buffor,
       warp_divergence_buffor,
       warp_size * n_emissions * sizeof(bool),
       hipMemcpyDeviceToHost);

  std::ofstream output;
  output.open("warp_info");

  for (int i = 0; i < warp_size * n_emissions; ++i) {

    output << int(cpu_warp_divergence_buffor[i]);
    if (i % warp_size == 0 && i != 0) {
      output << std::endl;
    }
  }

  delete cpu_warp_divergence_buffor;

#endif

  cuda(Memcpy,
       cpu_matrix,
       gpu_MatrixElement,
       n_tof_positions * number_of_blocks * sizeof(MatrixElement),
       hipMemcpyDeviceToHost);

#if NO_TOF > 0
  for (int lor_i = 0; lor_i < LORS; ++lor_i) {
    float temp = 0.f;
    for (int block_i = 0; block_i < number_of_blocks; ++block_i) {

      temp += cpu_matrix[block_i].hit[lor_i];
    }

    if (temp > 0.0f) {
      gpu_output[0].hit[lookup_table_lors[lor_i].index()] = temp;
    }
  }
#else
  for (int tof_i = 0; tof_i < n_tof_positions; ++tof_i) {
    for (int lor_i = 0; lor_i < LORS; ++lor_i) {
      float temp_hits = 0.f;
      for (int block_i = 0; block_i < number_of_blocks; ++block_i) {

        temp_hits += cpu_matrix[tof_i + (block_i * n_tof_positions)].hit[lor_i];
      }

      if (temp_hits > 0.0f) {

        gpu_output[tof_i].hit[lookup_table_lors[lor_i].index()] = temp_hits;
      }
    }
  }

#endif
  cuda(Free, gpu_prng_seed);
  cuda(Free, gpu_MatrixElement);

  return 0;
}
