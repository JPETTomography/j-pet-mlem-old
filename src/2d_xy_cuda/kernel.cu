// if we don't include that Qt Creator will show many errors
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include "config.h"
#include "data_structures.h"
#include "prng.cuh"
#include "geometry_methods.cuh"
#include "gpu_detector_geometry_test.cuh"
#include "gpu_detector_hit_test.cuh"
#include "gpu_phantom_generation.cuh"

static hipError_t err;

#define cuda(f, ...)                                        \
  if ((err = cuda##f(__VA_ARGS__)) != hipSuccess) {        \
    fprintf(stderr, #f "() %s\n", hipGetErrorString(err)); \
    exit(-1);                                               \
  }
#define cudathread_per_blockoSync(...) cuda(__VA_ARGS__)

double getwtime() {
  struct timeval tv;
  static time_t sec = 0;
  gettimeofday(&tv, NULL);
  if (!sec)
    sec = tv.tv_sec;
  return (double)(tv.tv_sec - sec) + (double)tv.tv_usec / 1e6;
}

void mem_clean_lors(Matrix_Element* cpu_matrix, int number_of_blocks) {

  for (int i = 0; i < number_of_blocks; ++i) {
    for (int j = 0; j < LORS; ++j) {

      cpu_matrix[i].lor[j] = 0.f;
    }
  }
}

void phantom_kernel(int number_of_threads_per_block,
                    int number_of_blocks,
                    int n_emissions,
                    int pixels_in_row,
                    float radius,
                    float h_detector,
                    float w_detector,
                    float pixel_size) {

  dim3 blocks(number_of_blocks);
  dim3 threads(number_of_threads_per_block);

  unsigned int* cpu_prng_seed;

  hipSetDevice(0);

  cpu_prng_seed =
      (unsigned int*)malloc(number_of_blocks * number_of_threads_per_block * 4 *
                            sizeof(unsigned int));

  for (int i = 0; i < 4 * number_of_blocks * number_of_threads_per_block; ++i) {

    cpu_prng_seed[i] = 53445 + i;
  }

  int triangular_matrix_size =
      ((pixels_in_row / 2) * ((pixels_in_row / 2) + 1) / 2);

  Matrix_Element* cpu_matrix =
      (Matrix_Element*)malloc(number_of_blocks * sizeof(Matrix_Element));

  // unsigned int matrix_size = triangular_matrix_size * number_of_blocks;

  unsigned int* gpu_prng_seed;
  Matrix_Element* gpu_matrix_element;

  cuda(Malloc,
       (void**)&gpu_prng_seed,
       number_of_blocks * number_of_threads_per_block * 4 *
           sizeof(unsigned int));
  cuda(Malloc,
       (void**)&gpu_matrix_element,
       number_of_blocks * sizeof(Matrix_Element));

  cuda(
      Memcpy,
      gpu_prng_seed,
      cpu_prng_seed,
      number_of_blocks * number_of_threads_per_block * 4 * sizeof(unsigned int),
      hipMemcpyHostToDevice);

  printf("GPU kernel start\n");
  printf(
      "Number of Detectors %d Numer of LORS: %d\n", NUMBER_OF_DETECTORS, LORS);

  double timer = getwtime();

  //  for (int j = pixels_in_row / 2 - 1; j >= 0; --j) {
  //    for (int i = 0; i <= j; ++i) {

  for (int j = 0; j < 1; ++j) {
    for (int i = 0; i < 1; ++i) {

      mem_clean_lors(cpu_matrix, number_of_blocks);

      cuda(Memcpy,
           gpu_matrix_element,
           cpu_matrix,
           number_of_blocks * sizeof(Matrix_Element),
           hipMemcpyHostToDevice);

      printf("Pixel(%d,%d) n_emissions: %d \n", i, j, n_emissions);

      gpu_phantom_generation << <blocks, threads>>>
          (i,
           j,
           n_emissions,
           gpu_prng_seed,
           gpu_matrix_element,
           number_of_threads_per_block,
           pixels_in_row,
           radius,
           h_detector,
           w_detector,
           pixel_size);

      hipDeviceSynchronize();

      cuda(Memcpy,
           cpu_matrix,
           gpu_matrix_element,
           number_of_blocks * sizeof(Matrix_Element),
           hipMemcpyDeviceToHost);

      for (int i = 0; i < LORS; i++) {
        float temp = 0.f;
        for (int j = 1; j < 2; ++j) {

          temp += cpu_matrix[j].lor[i];
        }

        if (temp > 0.0f) {
          printf("%f\n",
                 temp / number_of_blocks / number_of_threads_per_block /
                     n_emissions);
        }
      }
    }
  }
  double time = 0.0f;

  time = getwtime() - time;

  printf("time[s]: %f\n ", time);
  printf("time per pixel: %f\n", time / triangular_matrix_size);

  cuda(Free, gpu_prng_seed);
  cuda(Free, gpu_matrix_element);
}

void gpu_detector_geometry_kernel_test(float radius,
                                       float h_detector,
                                       float w_detector,
                                       float pixel_size,
                                       Detector_Ring& cpu_output) {

  dim3 blocks(1);
  dim3 threads(NUMBER_OF_DETECTORS);

  hipSetDevice(0);

  Detector_Ring* cpu_detectors = (Detector_Ring*)malloc(sizeof(Detector_Ring));

  Detector_Ring* gpu_detectors;

  cuda(Malloc, (void**)&gpu_detectors, sizeof(Detector_Ring));

  printf("Execute gpu_kernel_test for detectors geometry\n");

  cuda(Memcpy,
       gpu_detectors,
       cpu_detectors,
       sizeof(Detector_Ring),
       hipMemcpyHostToDevice);

  gpu_detector_geometry_test << <blocks, threads>>>
      (radius, h_detector, w_detector, pixel_size, gpu_detectors);

  hipDeviceSynchronize();

  cuda(Memcpy,
       cpu_detectors,
       gpu_detectors,
       sizeof(Detector_Ring),
       hipMemcpyDeviceToHost);

  for (int i = 0; i < NUMBER_OF_DETECTORS; ++i) {
    for (int j = 0; j < 4; j++) {

      cpu_output.detector_list[i].points[j].x =
          cpu_detectors->detector_list[i].points[j].x;
      cpu_output.detector_list[i].points[j].y =
          cpu_detectors->detector_list[i].points[j].y;
    }
  }

  cuda(Free, gpu_detectors);
}

void gpu_detector_hits_kernel_test(float crx,
                                   float cry,
                                   float cangle,
                                   float radius,
                                   float h_detector,
                                   float w_detector) {

  hipSetDevice(0);

  gpu_detector_hit_test << <1, NUMBER_OF_DETECTORS>>>
      (crx, cry, cangle, radius, h_detector, w_detector);

  hipDeviceSynchronize();
}
