#include "hip/hip_runtime.h"
// if we don't include that Qt Creator will show many errors
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include "config.h"
#include "data_structures.h"

/*-----------------------------------------GENERATORS-----------------------------------------------------*/

__constant__ unsigned int shift1[4] = { 6, 2, 13, 3 };
__constant__ unsigned int shift2[4] = { 13, 27, 21, 12 };
__constant__ unsigned int shift3[4] = { 18, 2, 7, 13 };
__constant__ unsigned int offset[4] = { 4294967294, 4294967288,
                                        4294967280, 4294967168 };

__shared__ unsigned int randStates[32];

//-----------end state rng
//---------------------------------------------------------//

__device__ float devData;

__device__ inline unsigned TausStep(unsigned& z,
                                    int S1,
                                    int S2,
                                    int S3,
                                    unsigned M) {
  unsigned b = (((z << S1) ^ z) >> S2);
  return z = (((z & M) << S3) ^ b);
}

__device__ inline unsigned LCGStep(unsigned& z, unsigned A, unsigned C) {
  return z = (A * z + C);
}

__device__ inline float HybridTaus(unsigned& z1,
                                   unsigned& z2,
                                   unsigned& z3,
                                   unsigned& z4) {
  return 2.3283064365387e-10f * (TausStep(z1, 14, 16, 15, 4294967294UL) ^
                                 TausStep(z2, 2, 44, 7, 4294967288UL) ^
                                 TausStep(z3, 3, 144, 17, 4294967280UL) ^
                                 LCGStep(z4, 1664525, 1013904223UL));
}

__device__ float rand_MWC_co(unsigned long long* x, unsigned int* a) {
  // Generate a random number [0,1)
  *x = (*x & 0xffffffffull) * (*a) + (*x >> 32);
  return __fdividef(
      __uint2float_rz((unsigned int)(*x)),
      (float)0x100000000);  // The typecast will truncate the x so that it is
                            // 0<=x<(2^32-1),__uint2float_rz ensures a round
                            // towards zero since 32-bit floating point cannot
                            // represent all integers that large. Dividing by
                            // 2^32 will hence yield [0,1)

}  // end __device__ rand_MWC_co

__device__ float rand_MWC_oc(unsigned long long* x, unsigned int* a) {
  // Generate a random number (0,1]
  return 1.0f - rand_MWC_co(x, a);
}  // end __device__ rand_MWC_oc

/*****************************METHODS*************************************************/

CUDA_CALLABLE_MEMBER Secant_Points
secant(float x, float y, float angle, float radius) {

  float a = std::sin(angle);
  float b = -std::cos(angle);
  float c = a * x + b * y;

  // std::cout << "a: " << a << " b: " << b << " c: " << c << std::endl;

  // helper variables
  float b2 = b * b;
  float b2c = b2 * c;
  float ac = a * c;
  float a2_b2 = a * a + b2;
  float b_a2_b2 = b * a2_b2;

  float sq = sqrt(b2 * (-(c * c) + a2_b2 * radius * radius));
  float asq = a * sq;

  // std::cout << "sq: " << sq << " asq: " << asq << std::endl;

  Secant_Points secant_positions;

  secant_positions.x1 = (ac - sq) / a2_b2;
  secant_positions.y1 = ((b2c + asq) / b_a2_b2);
  secant_positions.x2 = (ac + sq) / a2_b2;
  secant_positions.y2 = ((b2c - asq) / b_a2_b2);

  return secant_positions;
}

CUDA_CALLABLE_MEMBER Secant_Angle secant_angles(Secant_Points& e) {

  Secant_Angle temp;
  temp.angle1 = atan2(e.y1, e.x1);
  temp.angle2 = atan2(e.y2, e.x2);

  return temp;
}

CUDA_CALLABLE_MEMBER int section(float angle, int n_detectors) {
  // converting angles to [0,2 Pi) interval
  float normalised_angle = angle > 0 ? angle : (float)2.0 * M_PI + angle;
  return static_cast<int>(round(normalised_angle * n_detectors * ITP)) %
         (n_detectors);
}

CUDA_CALLABLE_MEMBER Secant_Sections
secant_sections(Secant_Points& e, int n_detectors) {

  Secant_Angle angles = secant_angles(e);

  Secant_Sections temp;

  temp.ss1 = section(angles.angle1, n_detectors);
  temp.ss2 = section(angles.angle2, n_detectors);

  return temp;
}

CUDA_CALLABLE_MEMBER int intersections(float x,
                                       float y,
                                       float angle,
                                       Detector_Ring& ring,
                                       int detector_id,
                                       Hits& hit) {

  // float temp_x =
  // ring.detector_list[detector_id].get_element(3).get_locationx();
  // float temp_y =
  // ring.detector_list[detector_id].get_element(3).get_locationy();

  float p1_x = ring.detector_list[detector_id].points[3].x;
  float p1_y = ring.detector_list[detector_id].points[3].y;

  // std::cout << "temp_x: " <<  temp_x << " " << "temp_y: " << temp_y <<
  // std::endl;

  float a = std::sin(angle);
  float b = -std::cos(angle);
  float c = a * x + b * y;

  // std::cout << "a: " << a << " " << "b: " << b << " " << "c: " << c <<
  // std::endl;

  float v1 = a * p1_x + b * p1_y - c;

  // std::cout <<"V1: " <<  v1 << std::endl;

  int r = 0;

  for (int i = 0; i < 4; i++) {

    float p2_x = ring.detector_list[detector_id].points[i].x;
    float p2_y = ring.detector_list[detector_id].points[i].y;

    float v2 = a * p2_x + b * p2_y - c;

    if (v2 == 0.0f) {
      hit.p[r].x = ring.detector_list[detector_id].points[i].x;
      hit.p[r].y = ring.detector_list[detector_id].points[i].y;
      // v2 is crossing point
      r++;
      // std::cout << " " << p2_x << "  " << p2_y;
      if (r == 2)
        return r;
    } else if (v1 * v2 < 0.0f) {
      // calculate intersection

      float m = a * (p1_x - p2_x) + b * (p1_y - p2_y);
      /*
       std::cout
       << (c * (p1_x - p2_x) + b * (p2_x * p1_y - p1_x * p2_y)) / m
       << "  "
       << ((c * (p1_y - p2_y) + a * (p1_x * p2_y - p2_x * p1_y))
       / m) << std::endl;*/
      hit.p[r].x = (c * (p1_x - p2_x) + b * (p2_x * p1_y - p1_x * p2_y)) / m;
      hit.p[r].y = (c * (p1_y - p2_y) + a * (p1_x * p2_y - p2_x * p1_y)) / m;

      r++;

      if (r == 2)
        return r;
    }
    v1 = v2;
    p1_x = p2_x;
    p1_y = p2_y;
  }
  return r;
}

CUDA_CALLABLE_MEMBER float secant_angle(float x1, float y1) {
  return atan2(y1, x1);
}

CUDA_CALLABLE_MEMBER bool check_for_hits(int inner,
                                         int outer,
                                         int x,
                                         int y,
                                         float angle,
                                         int n_detectors,
                                         Detector_Ring& ring,
                                         int& detector,
                                         Hits& hit) {

  int points;

  int step = ((n_detectors + inner - outer) % n_detectors >
              (n_detectors + outer - inner) % n_detectors)
                 ? 1
                 : n_detectors - 1;
  int end = (outer + step) % n_detectors;
  for (int i = inner; i != end; i = (i + step) % n_detectors) {
    points = intersections(x, y, angle, ring, i, hit);

    if (points == 2) {

      detector = i;
      return true;
    }

    // check if we got 2 point intersection
    // then test the model against these points distance
    // if (points.size() == 2) {
    //   auto deposition_depth = model.deposition_depth(gen);
    //   if (deposition_depth < (points[1] - points[0]).length()) {
    //    detector = i;
    //    depth = deposition_depth;
    //    return true;
    //  }
  }

  return false;
}

/***************************************************************************************/

static hipError_t err;

#define cuda(f, ...)                                        \
  if ((err = cuda##f(__VA_ARGS__)) != hipSuccess) {        \
    fprintf(stderr, #f "() %s\n", hipGetErrorString(err)); \
    exit(-1);                                               \
  }
#define cudathread_per_blockoSync(...) cuda(__VA_ARGS__)

double getwtime() {
  struct timeval tv;
  static time_t sec = 0;
  gettimeofday(&tv, NULL);
  if (!sec)
    sec = tv.tv_sec;
  return (double)(tv.tv_sec - sec) + (double)tv.tv_usec / 1e6;
}

__global__ void hello(char* a, int* b) {
  // increment chars with ints
  a[threadIdx.x] += b[threadIdx.x];
}

__device__ int lor_iterator(int& id1, int& id2) {

  if (id1 < id2) {
    int temp;
    temp = id2;
    id2 = id1;
    id1 = temp;
  }

  return ((id1 * (id1 + 1)) / 2) + id2;
}

__global__ void gpu_detector_geometry_test(float radius,
                                           float h_detector,
                                           float w_detector,
                                           float pixel_size,
                                           Detector_Ring* cpu_output) {

  __shared__ Detector_Ring test_ring;

  if (threadIdx.x < NUMBER_OF_DETECTORS) {

    Detectors detector_base;

    detector_base.points[0].x =
        (w_detector / 2.0f) + radius + (h_detector / 2.0f);
    detector_base.points[0].y = h_detector / 2.0f;
    detector_base.points[1].x =
        (w_detector / 2.0f) + radius + (h_detector / 2.0f);
    detector_base.points[1].y = -h_detector / 2.0f;
    detector_base.points[2].x =
        (-w_detector / 2.0f) + radius + (h_detector / 2.0f);
    detector_base.points[2].y = -h_detector / 2.0f;
    detector_base.points[3].x =
        (-w_detector / 2.0) + radius + (h_detector / 2.0f);
    detector_base.points[3].y = h_detector / 2.0f;

    test_ring.detector_list[threadIdx.x] = detector_base;

    float angle = 2.0f * M_PI * threadIdx.x / NUMBER_OF_DETECTORS;
    float sin_phi = __sinf(angle);
    float cos_phi = __cosf(angle);

    for (int j = 0; j < 4; ++j) {

      float temp_x = test_ring.detector_list[threadIdx.x].points[j].x;
      float temp_y = test_ring.detector_list[threadIdx.x].points[j].y;

      test_ring.detector_list[threadIdx.x].points[j].x =
          temp_x * cos_phi - temp_y * sin_phi;
      test_ring.detector_list[threadIdx.x].points[j].y =
          temp_x * sin_phi + temp_y * cos_phi;
    }
  }

  if (threadIdx.x < NUMBER_OF_DETECTORS) {

    cpu_output->detector_list[threadIdx.x] =
        test_ring.detector_list[threadIdx.x];
  }
}

__global__ void gpu_phantom_generation(int x,
                                       int y,
                                       int iteration,
                                       unsigned int* gpu_prng_seed,
                                       Matrix_Element* pixel_data,
                                       int threads,
                                       int pixels_in_row,
                                       float radius,
                                       float h_detector,
                                       float w_detector,
                                       float pixel_size) {

  int tid = ((blockIdx.x * blockDim.x) + threadIdx.x);

  unsigned int seed[4];

  seed[0] = gpu_prng_seed[4 * tid];
  seed[1] = gpu_prng_seed[4 * tid + 1];
  seed[2] = gpu_prng_seed[4 * tid + 2];
  seed[3] = gpu_prng_seed[4 * tid + 3];

  float inv_unit_prob_ = 0.1f;

  __shared__ Detector_Ring test_ring;

  Hits hit1;
  Hits hit2;

  float fov_radius = radius / M_SQRT2;

  if (threadIdx.x < NUMBER_OF_DETECTORS) {

    Detectors detector_base;

    detector_base.points[0].x =
        (w_detector / 2.0f) + radius + (h_detector / 2.0f);
    detector_base.points[0].y = h_detector / 2.0f;
    detector_base.points[1].x =
        (w_detector / 2.0f) + radius + (h_detector / 2.0f);
    detector_base.points[1].y = -h_detector / 2.0f;
    detector_base.points[2].x =
        (-w_detector / 2.0f) + radius + (h_detector / 2.0f);
    detector_base.points[2].y = -h_detector / 2.0f;
    detector_base.points[3].x =
        (-w_detector / 2.0) + radius + (h_detector / 2.0f);
    detector_base.points[3].y = h_detector / 2.0f;

    test_ring.detector_list[threadIdx.x] = detector_base;

    float angle = 2.0f * M_PI * threadIdx.x / NUMBER_OF_DETECTORS;
    float sin_phi = __sinf(angle);
    float cos_phi = __cosf(angle);

    for (int j = 0; j < 4; ++j) {

      float temp_x = test_ring.detector_list[threadIdx.x].points[j].x;
      float temp_y = test_ring.detector_list[threadIdx.x].points[j].y;

      test_ring.detector_list[threadIdx.x].points[j].x =
          temp_x * cos_phi - temp_y * sin_phi;
      test_ring.detector_list[threadIdx.x].points[j].y =
          temp_x * sin_phi + temp_y * cos_phi;
    }
  }

  __syncthreads();

  int detector1;
  int detector2;

  int i_inner;
  int i_outer;

#pragma unroll
  for (int i = 0; i < iteration; ++i) {

    if ((x * x + y * y) * pixel_size * pixel_size > fov_radius * fov_radius) {
      continue;
    }

    float rx =
        (x + HybridTaus(seed[0], seed[1], seed[2], seed[3])) * pixel_size;
    float ry =
        (y + HybridTaus(seed[0], seed[1], seed[2], seed[3])) * pixel_size;

    float angle = HybridTaus(seed[0], seed[1], seed[2], seed[3]) * M_PI;

    if (rx > ry) {
      continue;
    }

    // innetr and outer secant for circles
    Secant_Points inner_secant = secant(rx, ry, angle, radius);
    Secant_Points outer_secant = secant(rx, ry, angle, radius + h_detector);

    // hits per detector(if hits = 2 we got pair of detector, else generate
    // new random position and angle)

    i_inner = section(secant_angle(inner_secant.x1, inner_secant.y1),
                      NUMBER_OF_DETECTORS);
    i_outer = section(secant_angle(outer_secant.x1, inner_secant.y1),
                      NUMBER_OF_DETECTORS);

    if (!check_for_hits(i_inner,
                        i_outer,
                        rx,
                        ry,
                        angle,
                        NUMBER_OF_DETECTORS,
                        test_ring,
                        detector1,
                        hit1)) {
      continue;
    }

    i_inner = section(secant_angle(inner_secant.x2, inner_secant.y2),
                      NUMBER_OF_DETECTORS);
    i_outer = section(secant_angle(outer_secant.x2, inner_secant.y2),
                      NUMBER_OF_DETECTORS);

    if (!check_for_hits(i_inner,
                        i_outer,
                        rx,
                        ry,
                        angle,
                        NUMBER_OF_DETECTORS,
                        test_ring,
                        detector2,
                        hit2)) {
      continue;
    }

    float deposition_depth =
        -log(HybridTaus(seed[0], seed[1], seed[2], seed[3])) * inv_unit_prob_;

    // if(threadIdx.x == 1){
    // printf("TID:%d %d %d %d %d %f %f %f %f %f %f %f %f\n ", threadIdx.x, x,
    // y,
    //        detector1, detector2, hit1.p[0].x, hit1.p[0].y, hit1.p[1].x,
    //        hit1.p[1].y, hit2.p[0].x, hit2.p[0].y, hit2.p[1].x, hit2.p[1].y);
    // }
    // if(deposition_depth < (sqrt( (hit[1].x - hit[0].x) * (hit[1].x -
    // hit[0].x) + (hit[1].y - hit[0].y) * (hit[1].x - hit[0].x) ))) {

    atomicAdd(&pixel_data[blockIdx.x].lor[lor_iterator(detector1, detector2)],
              1.0f);
    //}
  }

  gpu_prng_seed[4 * tid] = seed[0];
  gpu_prng_seed[4 * tid + 1] = seed[1];
  gpu_prng_seed[4 * tid + 2] = seed[2];
  gpu_prng_seed[4 * tid + 3] = seed[3];
}

void run_kernel(char* str, int* val, int str_size, int val_size) {

  // CUDA side variables
  char* cuda_str;
  int* cuda_val;

  hipMalloc((void**)&cuda_str, str_size);
  hipMalloc((void**)&cuda_val, val_size);
  hipMemcpy(cuda_str, str, str_size, hipMemcpyHostToDevice);
  hipMemcpy(cuda_val, val, val_size, hipMemcpyHostToDevice);

  dim3 dimBlock(10, 1);
  dim3 dimGrid(1, 1);
  // Qt Creator does not like that, but ask NVIDIA about fancy notation
  // hello << <dimGrid, dimBlock>>> (cuda_str, cuda_val);

  hipMemcpy(str, cuda_str, str_size, hipMemcpyDeviceToHost);

  hipFree(cuda_str);
  hipFree(cuda_str);
}

void mem_clean_lors(Matrix_Element* cpu_matrix, int number_of_blocks) {

  for (int i = 0; i < number_of_blocks; ++i) {
    for (int j = 0; j < LORS; ++j) {

      cpu_matrix[i].lor[j] = 0.f;
    }
  }
}

void phantom_kernel(int number_of_threads_per_block,
                    int number_of_blocks,
                    int n_emissions,
                    int pixels_in_row,
                    float radius,
                    float h_detector,
                    float w_detector,
                    float pixel_size) {

  dim3 blocks(number_of_blocks);
  dim3 threads(number_of_threads_per_block);

  unsigned int* cpu_prng_seed;

  hipSetDevice(0);

  cpu_prng_seed =
      (unsigned int*)malloc(number_of_blocks * number_of_threads_per_block * 4 *
                            sizeof(unsigned int));

  for (int i = 0; i < 4 * number_of_blocks * number_of_threads_per_block; ++i) {

    cpu_prng_seed[i] = 53445 + i;
  }

  int triangular_matrix_size =
      ((pixels_in_row / 2) * ((pixels_in_row / 2) + 1) / 2);

  Matrix_Element* cpu_matrix =
      (Matrix_Element*)malloc(number_of_blocks * sizeof(Matrix_Element));

  // unsigned int matrix_size = triangular_matrix_size * number_of_blocks;

  unsigned int* gpu_prng_seed;
  Matrix_Element* gpu_matrix_element;

  cuda(Malloc,
       (void**)&gpu_prng_seed,
       number_of_blocks * number_of_threads_per_block * 4 *
           sizeof(unsigned int));
  cuda(Malloc,
       (void**)&gpu_matrix_element,
       number_of_blocks * sizeof(Matrix_Element));

  cuda(
      Memcpy,
      gpu_prng_seed,
      cpu_prng_seed,
      number_of_blocks * number_of_threads_per_block * 4 * sizeof(unsigned int),
      hipMemcpyHostToDevice);

  printf("GPU kernel start\n");
  printf(
      "Number of Detectors %d Numer of LORS: %d\n", NUMBER_OF_DETECTORS, LORS);

  double timer = getwtime();

  //  for (int j = pixels_in_row / 2 - 1; j >= 0; --j) {
  //    for (int i = 0; i <= j; ++i) {

  for (int j = 0; j < 1; ++j) {
    for (int i = 0; i < 1; ++i) {

      mem_clean_lors(cpu_matrix, number_of_blocks);

      cuda(Memcpy,
           gpu_matrix_element,
           cpu_matrix,
           number_of_blocks * sizeof(Matrix_Element),
           hipMemcpyHostToDevice);

      printf("Pixel(%d,%d) n_emissions: %d \n", i, j, n_emissions);

      gpu_phantom_generation << <blocks, threads>>>
          (i,
           j,
           n_emissions,
           gpu_prng_seed,
           gpu_matrix_element,
           number_of_threads_per_block,
           pixels_in_row,
           radius,
           h_detector,
           w_detector,
           pixel_size);

      hipDeviceSynchronize();

      cuda(Memcpy,
           cpu_matrix,
           gpu_matrix_element,
           number_of_blocks * sizeof(Matrix_Element),
           hipMemcpyDeviceToHost);

      for (int i = 0; i < LORS; i++) {
        float temp = 0.f;
        for (int j = 0; j < number_of_blocks; ++j) {

          temp += cpu_matrix[j].lor[i];
        }

        if (temp > 0.0f) {
          printf("%f\n", temp / number_of_blocks);
        }
      }
    }
  }
  double time = 0.0f;

  time = getwtime() - time;

  printf("time[s]: %f\n ", time);
  printf("time per pixel: %f\n", time / triangular_matrix_size);

  cuda(Free, gpu_prng_seed);
  cuda(Free, gpu_matrix_element);
}

void gpu_detector_geometry_kernel_test(float radius,
                                       float h_detector,
                                       float w_detector,
                                       float pixel_size,
                                       Detector_Ring& cpu_output) {

  dim3 blocks(1);
  dim3 threads(NUMBER_OF_DETECTORS);

  hipSetDevice(0);

  Detector_Ring* cpu_detectors = (Detector_Ring*)malloc(sizeof(Detector_Ring));

  Detector_Ring* gpu_detectors;

  cuda(Malloc, (void**)&gpu_detectors, sizeof(Detector_Ring));

  printf("Execute gpu_kernel_test for detectors geometry\n");

  cuda(Memcpy,
       gpu_detectors,
       cpu_detectors,
       sizeof(Detector_Ring),
       hipMemcpyHostToDevice);

  gpu_detector_geometry_test << <blocks, threads>>>
      (radius, h_detector, w_detector, pixel_size, gpu_detectors);

  hipDeviceSynchronize();

  cuda(Memcpy,
       cpu_detectors,
       gpu_detectors,
       sizeof(Detector_Ring),
       hipMemcpyDeviceToHost);

  for (int i = 0; i < NUMBER_OF_DETECTORS; ++i) {
    for (int j = 0; j < 4; j++) {

      cpu_output.detector_list[i].points[j].x =
          cpu_detectors->detector_list[i].points[j].x;
      cpu_output.detector_list[i].points[j].y =
          cpu_detectors->detector_list[i].points[j].y;
    }
  }

  cuda(Free, gpu_detectors);
}
