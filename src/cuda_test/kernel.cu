#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

static const int blocksize = 16;

__global__ void hello(char* a, int* b) { a[threadIdx.x] += b[threadIdx.x]; }

void run_kernel(char* str, int* val, int str_size, int val_size) {

  // CUDA side variables
  char* cuda_str;
  int* cuda_val;

  hipMalloc((void**)&cuda_str, str_size);
  hipMalloc((void**)&cuda_val, val_size);
  hipMemcpy(cuda_str, str, str_size, hipMemcpyHostToDevice);
  hipMemcpy(cuda_val, val, val_size, hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);
  hello << <dimGrid, dimBlock>>> (cuda_str, cuda_val);

  hipMemcpy(str, cuda_str, str_size, hipMemcpyDeviceToHost);

  hipFree(cuda_str);
  hipFree(cuda_str);
}
