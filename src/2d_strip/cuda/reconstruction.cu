#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "../event.h"
#include "../kernel.h"
#include "gpu_events_soa.h"
#include "config.h"

texture<float, 2, hipReadModeElementType> tex_sensitivity;
texture<float, 2, hipReadModeElementType> tex_rho;

#if THREAD_GRANULARITY
#include "reconstruction_thread_granularity.cuh"
#elif WARP_GRANULARITY
#include "reconstruction_warp_granularity.cuh"
#else
#include "reconstruction_simple.cuh"
#endif

template <typename F>
void fill_with_sensitivity(F* sensitivity,
                           F* inv_sensitivity,
                           StripDetector<F>& detector);

template <typename F>
void run_gpu_reconstruction(StripDetector<F>& detector,
                            Event<F>* events,
                            int n_events,
                            int n_iteration_blocks,
                            int n_iterations_in_block,
                            void (*output_callback)(StripDetector<F>& detector,
                                                    int iteration,
                                                    F* image,
                                                    void* context),
                            void (*progress_callback)(int iteration,
                                                      void* context),
                            void* context,
                            int device,
                            int n_blocks,
                            int n_threads_per_block,
                            bool verbose) {

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  if (verbose) {
    fprintf(stdout, "Running on: %s\n", prop.name);
  }

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#endif

  size_t image_size = detector.total_n_pixels * sizeof(F);

  const int width = detector.n_z_pixels;
  const int height = detector.n_y_pixels;

  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

  F* cpu_sensitivity = new F[detector.total_n_pixels];

  fill_with_sensitivity(cpu_sensitivity, detector);

  output_callback(detector, -1, cpu_sensitivity, context);

  F* gpu_sensitivity;
  size_t pitch_sensitivity;
  hipMallocPitch(
      &gpu_sensitivity, &pitch_sensitivity, sizeof(F) * width, height);
  hipMemcpy2D(gpu_sensitivity,
               pitch_sensitivity,
               cpu_sensitivity,
               sizeof(F) * width,
               sizeof(F) * width,
               height,
               hipMemcpyHostToDevice);
  delete[] cpu_sensitivity;

  hipBindTexture2D(NULL,
                    &tex_sensitivity,
                    gpu_sensitivity,
                    &desc,
                    width,
                    height,
                    pitch_sensitivity);

  F* cpu_rho = new F[detector.total_n_pixels];
  for (int i = 0; i < detector.total_n_pixels; ++i) {
    cpu_rho[i] = 100;
  }

  // this class allocated CUDA pointers and deallocated them in destructor
  GPUEventsSOA<F> gpu_events(events, n_events);

  F* gpu_rho;
  size_t pitch_rho;
  hipMallocPitch(&gpu_rho, &pitch_rho, sizeof(F) * width, height);
  hipBindTexture2D(NULL, &tex_rho, gpu_rho, &desc, width, height, pitch_rho);

  F* gpu_output_rho;

#if USE_RHO_PER_WARP
  hipMalloc((void**)&gpu_output_rho, n_blocks * image_size);
  F* cpu_output_rho;
  cpu_output_rho = new F[n_blocks * detector.total_n_pixels];
#else
  hipMalloc((void**)&gpu_output_rho, image_size);
#endif

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {

      hipEvent_t start, stop, start_mem_time, stop_mem_time;
      float time;
      float time_all;
      if (verbose) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventCreate(&start_mem_time);
        hipEventCreate(&stop_mem_time);
      } else {
        progress_callback(ib * n_iterations_in_block + it, context);
      }

#if USE_RHO_PER_WARP
      hipMemset(gpu_output_rho, 0, n_blocks * image_size);
#else
      hipMemset(gpu_output_rho, 0, image_size);
#endif
      hipMemcpy2D(gpu_rho,
                   pitch_rho,
                   cpu_rho,
                   sizeof(F) * width,
                   sizeof(F) * width,
                   height,
                   hipMemcpyHostToDevice);

      if (verbose) {
        hipEventRecord(start);
        hipEventRecord(start_mem_time);
      }
#ifdef __METRIC__
      F* gpu_metric_memory;
      const int metric_size = n_blocks * n_threads_per_block;
      hipMalloc((void**)&gpu_metric_memory, metric_size);
      hipMemset(gpu_metric_memory, 0, metric_size);
#endif

#if __HIPCC__
#define reconstruction reconstruction<Kernel> << <blocks, threads>>>
#endif
#ifdef __METRIC__
      reconstruction(gpu_metric_memory,
                     detector,
                     gpu_events.z_u,
                     gpu_events.z_d,
                     gpu_events.dl,
                     n_events,
                     gpu_output_rho,
                     n_blocks,
                     n_threads_per_block);
#else
      reconstruction(detector,
                     gpu_events.z_u,
                     gpu_events.z_d,
                     gpu_events.dl,
                     n_events,
                     gpu_output_rho,
                     n_blocks,
                     n_threads_per_block);
#endif

      hipDeviceSynchronize();

      if (verbose) {
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
      }

#ifdef __METRIC__
  hipFree(gpu_metric_memory);
#endif

#if USE_RHO_PER_WARP
      hipMemcpy(cpu_output_rho,
                 gpu_output_rho,
                 n_blocks * image_size,
                 hipMemcpyDeviceToHost);

      for (int i = 0; i < detector.n_y_pixels; ++i) {
        for (int j = 0; j < detector.n_z_pixels; ++j) {
          int pixel_adr = i * detector.n_y_pixels + j;
          cpu_rho[pixel_adr] = 0;
          for (int block_id = 0; block_id < n_blocks; ++block_id) {

            cpu_rho[i * detector.n_y_pixels + j] +=
                cpu_output_rho[block_id * detector.n_y_pixels + pixel_adr];
          }
        }
      }

#else
      hipMemcpy(cpu_rho, gpu_output_rho, image_size, hipMemcpyDeviceToHost);
#endif

      if (verbose) {
        hipEventRecord(stop_mem_time);
        hipEventSynchronize(stop_mem_time);
        hipEventElapsedTime(&time_all, start_mem_time, stop_mem_time);
        printf(
            "[%02d] kernel       : %f ms\n"
            "     kernel + mem : %f ms\n",
            ib * n_iterations_in_block + it,
            time,
            time_all);
      }
    }

    output_callback(detector, ib * n_iterations_in_block, cpu_rho, context);
  }

  if (!verbose) {
    progress_callback(n_iteration_blocks * n_iterations_in_block, context);
  }

  hipUnbindTexture(&tex_sensitivity);
  hipFree(gpu_sensitivity);
  hipUnbindTexture(&tex_rho);
  hipFree(gpu_rho);
  hipFree(gpu_output_rho);
  delete[] cpu_rho;
#if USE_RHO_PER_WARP
  delete[] cpu_output_rho;
#endif

}

template <typename F>
void fill_with_sensitivity(F* sensitivity, StripDetector<F>& detector) {

  size_t width = detector.n_z_pixels;
  size_t height = detector.n_y_pixels;

  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      sensitivity[y * width + x] = detector.pixel_sensitivity(Pixel<>(x, y));
    }
  }
}

// Explicit template instantiation

template void run_gpu_reconstruction<float>(
    StripDetector<float>& detector,
    Event<float>* events,
    int n_events,
    int n_iteration_blocks,
    int n_iterations_in_block,
    void (*output_callback)(StripDetector<float>& detector,
                            int iteration,
                            float* image,
                            void* context),
    void (*progress_callback)(int iteration, void* context),
    void* context,
    int device,
    int n_blocks,
    int n_threads_per_block,
    bool verbose);
