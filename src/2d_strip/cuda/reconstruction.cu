#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "../event.h"
#include "../kernel.h"
#include "config.h"

#if USE_SENSITIVITY
texture<float, 2, hipReadModeElementType> tex_inv_sensitivity;
#endif
texture<float, 2, hipReadModeElementType> tex_rho;

#if THREAD_GRANULARITY
#include "reconstruction_thread_granularity.cuh"
#elif WARP_GRANULARITY
#include "reconstruction_warp_granularity.cuh"
#else
#include "reconstruction_simple.cuh"
#endif

template <typename F>
void run_gpu_reconstruction(StripDetector<F>& detector,
                            Event<F>* events,
                            int n_events,
                            int n_iteration_blocks,
                            int n_iterations_in_block,
                            void (*output_callback)(StripDetector<F>& detector,
                                                    int iteration,
                                                    F* image,
                                                    void* context),
                            void (*progress_callback)(int iteration,
                                                      void* context),
                            void* context,
                            int device,
                            int n_blocks,
                            int n_threads_per_block,
                            bool verbose) {

  hipSetDevice(device);

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#endif

  size_t image_size = detector.total_n_pixels * sizeof(F);
  size_t events_size = n_events * sizeof(F);

  const int width = detector.n_z_pixels;
  const int height = detector.n_y_pixels;

#if USE_SENSITIVITY
  F* cpu_inv_sensitivity = (F*)malloc(image_size);
  F* cpu_sensitivity = (F*)malloc(image_size);
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      Point<F> point = detector.pixel_center(Pixel<>(x, y));
      F pixel_sensitivity = detector.sensitivity(point);
      cpu_sensitivity[y * width + x] = pixel_sensitivity;
      cpu_inv_sensitivity[y * width + x] = 1 / pixel_sensitivity;
    }
  }

  output_callback(detector, -1, cpu_sensitivity, context);
  free(cpu_sensitivity);
#endif

  F* cpu_rho = (F*)malloc(image_size);

  for (int i = 0; i < detector.total_n_pixels; ++i) {
    cpu_rho[i] = 100;
  }

  F* cpu_events_z_u = (F*)malloc(events_size);
  F* cpu_events_z_d = (F*)malloc(events_size);
  F* cpu_events_dl = (F*)malloc(events_size);

  for (int i = 0; i < n_events; ++i) {
    cpu_events_z_u[i] = events[i].z_u;
    cpu_events_z_d[i] = events[i].z_d;
    cpu_events_dl[i] = events[i].dl;
  }

  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

#if USE_SENSITIVITY
  F* gpu_inv_sensitivity;
  size_t pitch_inv_sensitivity;
  hipMallocPitch(
      &gpu_inv_sensitivity, &pitch_inv_sensitivity, sizeof(F) * width, height);
  hipMemcpy2D(gpu_inv_sensitivity,
               pitch_inv_sensitivity,
               cpu_inv_sensitivity,
               sizeof(F) * width,
               sizeof(F) * width,
               height,
               hipMemcpyHostToDevice);
  free(cpu_inv_sensitivity);
  hipBindTexture2D(NULL,
                    &tex_inv_sensitivity,
                    gpu_inv_sensitivity,
                    &desc,
                    width,
                    height,
                    pitch_inv_sensitivity);
#endif

  F* gpu_rho;
  size_t pitch_rho;
  hipMallocPitch(&gpu_rho, &pitch_rho, sizeof(F) * width, height);
  hipBindTexture2D(NULL, &tex_rho, gpu_rho, &desc, width, height, pitch_rho);

  F* gpu_output_rho;

#if USE_WARP_IMAGE_SPACE
  hipMalloc((void**)&gpu_output_rho, n_blocks * image_size);
  F* cpu_output_rho;
  cpu_output_rho = (F*)malloc(n_blocks * image_size);
#else
  hipMalloc((void**)&gpu_output_rho, image_size);
#endif

  F* gpu_events_z_u;
  F* gpu_events_z_d;
  F* gpu_events_dl;

  hipMalloc((void**)&gpu_events_z_u, events_size);
  hipMalloc((void**)&gpu_events_z_d, events_size);
  hipMalloc((void**)&gpu_events_dl, events_size);

  hipMemcpy(
      gpu_events_z_u, cpu_events_z_u, events_size, hipMemcpyHostToDevice);
  hipMemcpy(
      gpu_events_z_d, cpu_events_z_d, events_size, hipMemcpyHostToDevice);
  hipMemcpy(gpu_events_dl, cpu_events_dl, events_size, hipMemcpyHostToDevice);

  free(cpu_events_z_u);
  free(cpu_events_z_d);
  free(cpu_events_dl);

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {

      hipEvent_t start, stop, start_mem_time, stop_mem_time;
      float time;
      float time_all;
      if (verbose) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventCreate(&start_mem_time);
        hipEventCreate(&stop_mem_time);
      } else {
        progress_callback(ib * n_iterations_in_block + it, context);
      }

#if USE_WARP_IMAGE_SPACE
      hipMemset(gpu_output_rho, 0, n_blocks * image_size);
#else
      hipMemset(gpu_output_rho, 0, image_size);
#endif
      hipMemcpy2D(gpu_rho,
                   pitch_rho,
                   cpu_rho,
                   sizeof(F) * width,
                   sizeof(F) * width,
                   height,
                   hipMemcpyHostToDevice);

      if (verbose) {
        hipEventRecord(start);
        hipEventRecord(start_mem_time);
      }

#if __HIPCC__
#define reconstruction reconstruction<Kernel> << <blocks, threads>>>
#endif
      reconstruction(detector,
                     gpu_events_z_u,
                     gpu_events_z_d,
                     gpu_events_dl,
                     n_events,
                     gpu_output_rho,
                     n_blocks,
                     n_threads_per_block);

      hipDeviceSynchronize();

      if (verbose) {
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
      }

#if USE_WARP_IMAGE_SPACE
      hipMemcpy(cpu_output_rho,
                 gpu_output_rho,
                 n_blocks * image_size,
                 hipMemcpyDeviceToHost);

      for (int i = 0; i < detector.n_y_pixels; ++i) {
        for (int j = 0; j < detector.n_z_pixels; ++j) {
          int pixel_adr = i * detector.n_y_pixels + j;
          cpu_rho[pixel_adr] = 0;
          for (int block_id = 0; block_id < n_blocks; ++block_id) {

            cpu_rho[i * detector.n_y_pixels + j] +=
                cpu_output_rho[block_id * detector.n_y_pixels + pixel_adr];
          }
        }
      }

#else
      hipMemcpy(cpu_rho, gpu_output_rho, image_size, hipMemcpyDeviceToHost);
#endif

      if (verbose) {
        hipEventRecord(stop_mem_time);
        hipEventSynchronize(stop_mem_time);
        hipEventElapsedTime(&time_all, start_mem_time, stop_mem_time);
        printf(
            "[%02d] kernel       : %f ms\n"
            "     kernel + mem : %f ms\n",
            ib * n_iterations_in_block + it,
            time,
            time_all);
      }
    }

    output_callback(detector, ib * n_iterations_in_block, cpu_rho, context);
  }

  if (!verbose) {
    progress_callback(n_iteration_blocks * n_iterations_in_block, context);
  }

#if USE_SENSITIVITY
  hipUnbindTexture(&tex_inv_sensitivity);
  hipFree(gpu_inv_sensitivity);
#endif
  hipUnbindTexture(&tex_rho);
  hipFree(gpu_rho);
  hipFree(gpu_events_z_u);
  hipFree(gpu_events_z_d);
  hipFree(gpu_events_dl);
  hipFree(gpu_output_rho);
  free(cpu_rho);
#if USE_WARP_IMAGE_SPACE
  free(cpu_output_rho);
#endif
}

template void run_gpu_reconstruction<float>(
    StripDetector<float>& detector,
    Event<float>* events,
    int n_events,
    int n_iteration_blocks,
    int n_iterations_in_block,
    void (*output_callback)(StripDetector<float>& detector,
                            int iteration,
                            float* image,
                            void* context),
    void (*progress_callback)(int iteration, void* context),
    void* context,
    int device,
    int n_blocks,
    int n_threads_per_block,
    bool verbose);
