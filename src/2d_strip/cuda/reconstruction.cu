#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "../event.h"

#include "config.h"

#if EVENT_GRANULARITY
#include "reconstruction_event_granularity.cuh"
#elif WARP_GRANULARITY
#include "reconstruction_warp_granularity.cuh"
#else
#include "reconstruction_simple.cuh"
#endif

template <typename F>
void run_gpu_reconstruction(StripDetector<F>& detector,
                            Event<F>* events,
                            int n_events,
                            int n_iteration_blocks,
                            int n_iterations_in_block,
                            void (*output_callback)(StripDetector<F>& detector,
                                                    int iteration,
                                                    F* image,
                                                    void* context),
                            void (*progress_callback)(int iteration,
                                                      void* context),
                            void* context,
                            int device,
                            int n_blocks,
                            int n_threads_per_block) {

  hipSetDevice(device);

  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);

  size_t image_size = detector.total_n_pixels * sizeof(F);
  size_t output_size = image_size * n_blocks;
  size_t events_size = n_events * sizeof(Event<F>);

  F* cpu_output = (F*)malloc(image_size);
  F* cpu_rho = (F*)malloc(image_size);
  F* cpu_sensitivity = (F*)malloc(image_size);

  for (int i = 0; i < detector.total_n_pixels; ++i) {
    cpu_rho[i] = 100;
  }

  for (int x = 0; x < detector.n_y_pixels; ++x) {
    for (int y = 0; y < detector.n_z_pixels; ++y) {
      Point<F> point = detector.pixel_center(x, y);
      cpu_sensitivity[x * detector.n_z_pixels + y] =
          detector.sensitivity(point.x, point.y);
    }
  }

  F* gpu_output;
  F* gpu_rho;
  Event<F>* gpu_events;
  F* gpu_soa_events;
  F* cpu_soa_events;

  cpu_soa_events = (F*)malloc(events_size);
  for (int i = 0; i < n_events; ++i) {
    cpu_soa_events[i + 0 * n_events] = events[i].z_u;
    cpu_soa_events[i + 1 * n_events] = events[i].z_d;
    cpu_soa_events[i + 2 * n_events] = events[i].dl;
  }

  // declare and allocate memory
  F* gpu_sensitivity;

  size_t pitch;
  hipMallocPitch(&gpu_sensitivity,
                  &pitch,
                  sizeof(F) * detector.n_y_pixels,
                  detector.n_z_pixels);

  hipMemcpy2D(gpu_sensitivity,
               pitch,
               cpu_sensitivity,
               sizeof(F) * detector.n_y_pixels,
               sizeof(F) * detector.n_y_pixels,
               detector.n_z_pixels,
               hipMemcpyHostToDevice);

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = gpu_sensitivity;
  resDesc.res.pitch2D.pitchInBytes = pitch;
  resDesc.res.pitch2D.width = detector.n_y_pixels;
  resDesc.res.pitch2D.height = detector.n_z_pixels;
  // resDesc.res.pitch2D.desc = hipCreateChannelDesc<F>();
  resDesc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
  resDesc.res.pitch2D.desc.x = 32;  // 32 bits per channel for float texture
  resDesc.res.pitch2D.desc.y = 0;   // set 32 bits ONLY for float2 texture
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex_sensitivity;
  hipCreateTextureObject(&tex_sensitivity, &resDesc, &texDesc, NULL);

  hipMalloc((void**)&gpu_soa_events, events_size);
  hipMemcpy(
      gpu_soa_events, cpu_soa_events, events_size, hipMemcpyHostToDevice);

  hipMalloc((void**)&gpu_events, events_size);
  hipMemcpy(gpu_events, events, events_size, hipMemcpyHostToDevice);

  hipMalloc((void**)&gpu_output, output_size);
  hipMalloc((void**)&gpu_rho, image_size);

  output_callback(detector, -1, cpu_sensitivity, context);

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {
      progress_callback(ib * n_iterations_in_block + it, context);

      hipMemset(gpu_output, 0, output_size);
      hipMemcpy(gpu_rho, cpu_rho, image_size, hipMemcpyHostToDevice);

      reconstruction_2d_strip_cuda<F> << <blocks, threads>>>
          (detector,
           gpu_soa_events,
           n_events,
           gpu_output,
           gpu_rho,
           tex_sensitivity,
           n_blocks,
           n_threads_per_block);

      hipDeviceSynchronize();

      // grab output
      hipMemcpy(cpu_output,
                 gpu_output,
                 image_size * n_blocks,
                 hipMemcpyDeviceToHost);

      // merge image output from all blocks
      for (int block = 0; block < n_blocks; ++block) {
        for (int p = 0; p < detector.total_n_pixels; ++p) {
          cpu_rho[p] += cpu_output[block * detector.total_n_pixels + p];
        }
      }
    }

    output_callback(detector, ib * n_iterations_in_block, cpu_rho, context);
  }

  progress_callback(n_iteration_blocks * n_iterations_in_block, context);

  hipDestroyTextureObject(tex_sensitivity);
  hipFree(gpu_soa_events);
  hipFree(gpu_events);
  hipFree(gpu_output);
  hipFree(gpu_rho);
  hipFree(gpu_sensitivity);
  free(cpu_soa_events);
  free(cpu_output);
  free(cpu_rho);
  free(cpu_sensitivity);
}

template void run_gpu_reconstruction<float>(
    StripDetector<float>& detector,
    Event<float>* events,
    int n_events,
    int n_iteration_blocks,
    int n_iterations_in_block,
    void (*output_callback)(StripDetector<float>& detector,
                            int iteration,
                            float* image,
                            void* context),
    void (*progress_callback)(int iteration, void* context),
    void* context,
    int device,
    int n_blocks,
    int n_threads_per_block);
