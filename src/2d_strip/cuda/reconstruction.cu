#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

#include "util/cuda/debug.h"  // catches all CUDA errors
#include "../event.h"
#include "../kernel.h"
#include "gpu_events_soa.h"
#include "config.h"

#if USE_SENSITIVITY
texture<float, 2, hipReadModeElementType> tex_inv_sensitivity;
#endif
texture<float, 2, hipReadModeElementType> tex_rho;

#if THREAD_GRANULARITY
#include "reconstruction_thread_granularity.cuh"
#elif WARP_GRANULARITY
#include "reconstruction_warp_granularity.cuh"
#else
#include "reconstruction_simple.cuh"
#endif

template <typename F>
void fill_with_sensitivity(F* sensitivity,
                           F* inv_sensitivity,
                           StripDetector<F>& detector);

template <typename F>
void run_gpu_reconstruction(StripDetector<F>& detector,
                            Event<F>* events,
                            int n_events,
                            int n_iteration_blocks,
                            int n_iterations_in_block,
                            void (*output_callback)(StripDetector<F>& detector,
                                                    int iteration,
                                                    F* image,
                                                    void* context),
                            void (*progress_callback)(int iteration,
                                                      void* context),
                            void* context,
                            int device,
                            int n_blocks,
                            int n_threads_per_block,
                            bool verbose) {

  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  fprintf(stdout, "Running on %s", prop.name);

#if __HIPCC__
  dim3 blocks(n_blocks);
  dim3 threads(n_threads_per_block);
#endif

  size_t image_size = detector.total_n_pixels * sizeof(F);

  const int width = detector.n_z_pixels;
  const int height = detector.n_y_pixels;

  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

#if USE_SENSITIVITY
  F* cpu_inv_sensitivity = new F[detector.total_n_pixels];
  F* cpu_sensitivity = new F[detector.total_n_pixels];

  fill_with_sensitivity(cpu_sensitivity, cpu_inv_sensitivity, detector);

  output_callback(detector, -1, cpu_sensitivity, context);
  delete[] cpu_sensitivity;
  F* gpu_inv_sensitivity;
  size_t pitch_inv_sensitivity;
  hipMallocPitch(
      &gpu_inv_sensitivity, &pitch_inv_sensitivity, sizeof(F) * width, height);
  hipMemcpy2D(gpu_inv_sensitivity,
               pitch_inv_sensitivity,
               cpu_inv_sensitivity,
               sizeof(F) * width,
               sizeof(F) * width,
               height,
               hipMemcpyHostToDevice);
  delete[] cpu_inv_sensitivity;

  hipBindTexture2D(NULL,
                    &tex_inv_sensitivity,
                    gpu_inv_sensitivity,
                    &desc,
                    width,
                    height,
                    pitch_inv_sensitivity);

#endif

  F* cpu_rho = new F[detector.total_n_pixels];
  std::fill_n(cpu_rho, F(100), detector.total_n_pixels);

  // this class allocated CUDA pointers and deallocated them in destructor
  GPUEventsSOA<F> gpu_events(events, n_events);

  F* gpu_rho;
  size_t pitch_rho;
  hipMallocPitch(&gpu_rho, &pitch_rho, sizeof(F) * width, height);
  hipBindTexture2D(NULL, &tex_rho, gpu_rho, &desc, width, height, pitch_rho);

  F* gpu_output_rho;

#if USE_WARP_IMAGE_SPACE
  hipMalloc((void**)&gpu_output_rho, n_blocks * image_size);
  F* cpu_output_rho;
  cpu_output_rho = new F[n_blocks * detector.total_n_pixels];
#else
  hipMalloc((void**)&gpu_output_rho, image_size);
#endif

  for (int ib = 0; ib < n_iteration_blocks; ++ib) {
    for (int it = 0; it < n_iterations_in_block; ++it) {

      hipEvent_t start, stop, start_mem_time, stop_mem_time;
      float time;
      float time_all;
      if (verbose) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventCreate(&start_mem_time);
        hipEventCreate(&stop_mem_time);
      } else {
        progress_callback(ib * n_iterations_in_block + it, context);
      }

#if USE_WARP_IMAGE_SPACE
      hipMemset(gpu_output_rho, 0, n_blocks * image_size);
#else
      hipMemset(gpu_output_rho, 0, image_size);
#endif
      hipMemcpy2D(gpu_rho,
                   pitch_rho,
                   cpu_rho,
                   sizeof(F) * width,
                   sizeof(F) * width,
                   height,
                   hipMemcpyHostToDevice);

      if (verbose) {
        hipEventRecord(start);
        hipEventRecord(start_mem_time);
      }

#if __HIPCC__
#define reconstruction reconstruction<Kernel> << <blocks, threads>>>
#endif
      reconstruction(detector,
                     gpu_events.z_u,
                     gpu_events.z_d,
                     gpu_events.dl,
                     n_events,
                     gpu_output_rho,
                     n_blocks,
                     n_threads_per_block);

      hipDeviceSynchronize();

      if (verbose) {
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
      }

#if USE_WARP_IMAGE_SPACE
      hipMemcpy(cpu_output_rho,
                 gpu_output_rho,
                 n_blocks * image_size,
                 hipMemcpyDeviceToHost);

      for (int i = 0; i < detector.n_y_pixels; ++i) {
        for (int j = 0; j < detector.n_z_pixels; ++j) {
          int pixel_adr = i * detector.n_y_pixels + j;
          cpu_rho[pixel_adr] = 0;
          for (int block_id = 0; block_id < n_blocks; ++block_id) {

            cpu_rho[i * detector.n_y_pixels + j] +=
                cpu_output_rho[block_id * detector.n_y_pixels + pixel_adr];
          }
        }
      }

#else
      hipMemcpy(cpu_rho, gpu_output_rho, image_size, hipMemcpyDeviceToHost);
#endif

      if (verbose) {
        hipEventRecord(stop_mem_time);
        hipEventSynchronize(stop_mem_time);
        hipEventElapsedTime(&time_all, start_mem_time, stop_mem_time);
        printf(
            "[%02d] kernel       : %f ms\n"
            "     kernel + mem : %f ms\n",
            ib * n_iterations_in_block + it,
            time,
            time_all);
      }
    }

    output_callback(detector, ib * n_iterations_in_block, cpu_rho, context);
  }

  if (!verbose) {
    progress_callback(n_iteration_blocks * n_iterations_in_block, context);
  }

#if USE_SENSITIVITY
  hipUnbindTexture(&tex_inv_sensitivity);
  hipFree(gpu_inv_sensitivity);
#endif
  hipUnbindTexture(&tex_rho);
  hipFree(gpu_rho);
  hipFree(gpu_output_rho);
  delete[] cpu_rho;
#if USE_WARP_IMAGE_SPACE
  delete[] cpu_output_rho;
#endif
}

template void run_gpu_reconstruction<float>(
    StripDetector<float>& detector,
    Event<float>* events,
    int n_events,
    int n_iteration_blocks,
    int n_iterations_in_block,
    void (*output_callback)(StripDetector<float>& detector,
                            int iteration,
                            float* image,
                            void* context),
    void (*progress_callback)(int iteration, void* context),
    void* context,
    int device,
    int n_blocks,
    int n_threads_per_block,
    bool verbose);

template <typename F>
void fill_with_sensitivity(F* sensitivity,
                           F* inv_sensitivity,
                           StripDetector<F>& detector) {

  size_t width = detector.n_z_pixels;
  size_t height = detector.n_y_pixels;

  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      Point<F> point = detector.pixel_center(Pixel<>(x, y));
      F pixel_sensitivity = detector.sensitivity(point);
      sensitivity[y * width + x] = pixel_sensitivity;
      inv_sensitivity[y * width + x] = 1 / pixel_sensitivity;
    }
  }
}
