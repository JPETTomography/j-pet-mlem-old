#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "reconstruction.cuh"

static hipError_t err;

#define cuda_kernel_config(blocks, threads)                      \
  {                                                              \
    printf("Cuda kernel config\n");                              \
    printf("Number of  blocks per kernel: %d\n", blocks);        \
    printf("Number of threads|block per kernel: %d\n", threads); \
  }

#define cuda(f, ...)                                        \
  if ((err = cuda##f(__VA_ARGS__)) != hipSuccess) {        \
    fprintf(stderr, #f "() %s\n", hipGetErrorString(err)); \
    exit(-1);                                               \
  }

#define cudathread_per_blockoSync(...) cuda(__VA_ARGS__)

void gpu_reconstruction_strip_2d(CUDA::Config cfg,
                                 Event<float>* event_list,
                                 int event_size,
                                 int iteration_chunk,
                                 float* image_output,
                                 int off) {

  hipSetDevice(0);

  dim3 blocks(cfg.number_of_blocks);
  dim3 threads(cfg.number_of_threads_per_block);

  // cuda_kernel_config(cfg.number_of_blocks,
  // cfg.number_of_threads_per_block);

  size_t image_sz = cfg.n_pixels * cfg.n_pixels * sizeof(float);

  float* cpu_image_buffor = (float*)malloc(image_sz * cfg.number_of_blocks);

  float* cpu_image_rho = (float*)malloc(image_sz);

  float* cpu_temp_rho = (float*)malloc(image_sz);

  float cpu_image_sensitivity[image_sz];

  for (int i = 0; i < cfg.n_pixels * cfg.n_pixels; ++i) {

    cpu_image_rho[i] = 100.0f;
  }

  for (int i = 0; i < cfg.number_of_blocks * cfg.n_pixels * cfg.n_pixels; ++i) {

    cpu_image_buffor[i] = 0.f;
  }

  float half_pixel_size = 0.5 * cfg.pixel_size;
  float half_grid_size = 0.5f * cfg.grid_size_y_;

  for (int px = 0; px < cfg.n_pixels; ++px) {
    for (int py = 0; py < cfg.n_pixels; ++py) {

      float2 pixel_coordiantes = pixel_center(px,
                                              py,
                                              cfg.pixel_size,
                                              cfg.pixel_size,
                                              cfg.grid_size_y_,
                                              cfg.grid_size_z_,
                                              half_grid_size,
                                              half_pixel_size);

      cpu_image_sensitivity[px * cfg.n_pixels + py] =
          sensitivity(pixel_coordiantes.x,
                      pixel_coordiantes.y,
                      cfg.R_distance,
                      cfg.Scentilator_length / 2.0f);
    }
  }

  float* gpu_image_buffor;
  float* gpu_image_rho;
  Event<float>* gpu_event_list;
  soa_event<float>* gpu_soa_event_list;

  soa_event<float>* cpu_soa_event_list;

  cpu_soa_event_list = (soa_event<float>*)malloc(sizeof(soa_event<float>));

#ifdef OFFSET_WARP_TEST

  int offset = off;

  event<float> data_chunk[offset];

  for (int i = 0; i < offset; ++i) {

    data_chunk[i] = event_list[i];
  }

  cpu_soa_event_list->set_data_chunk(data_chunk, offset, event_size);

#else

  cpu_soa_event_list->set_data(event_list, event_size);

#endif
  // declare and allocate memory
  float* texture_sensitivity_buffer;

  size_t pitch;
  hipMallocPitch(&texture_sensitivity_buffer,
                  &pitch,
                  sizeof(float) * cfg.n_pixels,
                  cfg.n_pixels);

  hipMemcpy2D(texture_sensitivity_buffer,
               pitch,
               &cpu_image_sensitivity,
               sizeof(float) * cfg.n_pixels,
               sizeof(float) * cfg.n_pixels,
               cfg.n_pixels,
               hipMemcpyHostToDevice);

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = texture_sensitivity_buffer;
  resDesc.res.pitch2D.pitchInBytes = pitch;
  resDesc.res.pitch2D.width = cfg.n_pixels;
  resDesc.res.pitch2D.height = cfg.n_pixels;
  // resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
  resDesc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
  resDesc.res.pitch2D.desc.x = 32;  // 32 bits per channel for float texture
  resDesc.res.pitch2D.desc.y = 0;   // set 32 bits ONLY for float2 texture
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  // other mallocs and allocations

  cuda(Malloc, (void**)&gpu_event_list, event_size * sizeof(Event<float>));
  cuda(Malloc, (void**)&gpu_image_buffor, image_sz * cfg.number_of_blocks);
  cuda(Malloc, (void**)&gpu_image_rho, image_sz);
  cuda(Malloc, (void**)&gpu_soa_event_list, sizeof(soa_event<float>));

  cuda(Memcpy,
       gpu_soa_event_list,
       cpu_soa_event_list,
       sizeof(soa_event<float>),
       hipMemcpyHostToDevice);

  cuda(Memcpy,
       gpu_event_list,
       event_list,
       event_size * sizeof(Event<float>),
       hipMemcpyHostToDevice);

  cuda(Memcpy,
       gpu_image_buffor,
       cpu_image_buffor,
       image_sz * cfg.number_of_blocks,
       hipMemcpyHostToDevice);

  cuda(Memcpy, gpu_image_rho, cpu_image_rho, image_sz, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  for (int i = 0; i < iteration_chunk; ++i) {

    reconstruction_2d_strip_cuda<float> << <blocks, threads>>>
        (cfg,
         gpu_soa_event_list,
         gpu_event_list,
         event_size,
         gpu_image_buffor,
         gpu_image_rho,
         tex);

    hipDeviceSynchronize();

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time: %f\n", milliseconds / 1000);

    //    unsigned int number_of_ops_per_kernel = 44921;

    cuda(Memcpy,
         cpu_image_buffor,
         gpu_image_buffor,
         image_sz * cfg.number_of_blocks,
         hipMemcpyDeviceToHost);

    for (int block_id = 0; block_id < cfg.number_of_blocks; ++block_id) {
      for (int index = 0; index < cfg.n_pixels * cfg.n_pixels; ++index) {

        image_output[(i * cfg.n_pixels * cfg.n_pixels) + index] +=
            cpu_image_buffor[block_id * cfg.n_pixels * cfg.n_pixels + index];
      }
    }

    for (int pixel = 0;
         pixel < cfg.number_of_blocks * cfg.n_pixels * cfg.n_pixels;
         ++pixel) {

      cpu_image_buffor[pixel] = 0.f;
    }

    for (int pixel = 0; pixel < cfg.n_pixels * cfg.n_pixels; ++pixel) {

      cpu_temp_rho[pixel] =
          image_output[(i * cfg.n_pixels * cfg.n_pixels) + pixel];
    }

    cuda(Memcpy,
         gpu_image_buffor,
         cpu_image_buffor,
         image_sz * cfg.number_of_blocks,
         hipMemcpyHostToDevice);

    cuda(Memcpy, gpu_image_rho, cpu_temp_rho, image_sz, hipMemcpyHostToDevice);
  }

  // clean heap
  cuda(DestroyTextureObject, tex);
  cuda(Free, gpu_image_buffor);
  cuda(Free, gpu_image_rho);
  cuda(Free, texture_sensitivity_buffer);
  cuda(Free, gpu_soa_event_list);
  free(cpu_temp_rho);
  free(cpu_image_buffor);
  free(cpu_image_rho);
  free(cpu_soa_event_list);
}
