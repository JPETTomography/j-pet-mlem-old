#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "reconstruction.cuh"

static hipError_t err;

#define cuda_kernel_config(blocks, threads)                      \
  {                                                              \
    printf("Cuda kernel config\n");                              \
    printf("Number of  blocks per kernel: %d\n", blocks);        \
    printf("Number of threads|block per kernel: %d\n", threads); \
  }

#define cuda(f, ...)                                        \
  if ((err = cuda##f(__VA_ARGS__)) != hipSuccess) {        \
    fprintf(stderr, #f "() %s\n", hipGetErrorString(err)); \
    exit(-1);                                               \
  }
#define cudathread_per_blockoSync(...) cuda(__VA_ARGS__)

void gpu_reconstruction_strip_2d(gpu_config::GPU_parameters cfg,
                                 event<float>* event_list,
                                 int event_size,
                                 int iteration_chunk,
                                 float* image_output) {

  hipSetDevice(0);

  printf("Data Size: %d \n", event_size);

  dim3 blocks(cfg.number_of_blocks);
  dim3 threads(cfg.number_of_threads_per_block);

  cuda_kernel_config(cfg.number_of_blocks, cfg.number_of_threads_per_block);

  size_t image_sz = cfg.n_pixels * cfg.n_pixels * sizeof(float);

  float* cpu_image_buffor = (float*)malloc(image_sz * cfg.number_of_blocks);

  float* cpu_image_rho = (float*)malloc(image_sz);

  float cpu_image_sensitivity[image_sz];

  for (int i = 0; i < cfg.n_pixels * cfg.n_pixels; ++i) {

    cpu_image_rho[i] = 100.0f;
  }

  for (int i = 0; i < cfg.number_of_blocks * cfg.n_pixels * cfg.n_pixels; ++i) {

    cpu_image_buffor[i] = 0.f;
  }

  for (int px = 0; px < cfg.n_pixels; ++px) {
    for (int py = 0; py < cfg.n_pixels; ++py) {

      float2 pixel_coordiantes = pixel_center(px,
                                              py,
                                              cfg.pixel_size,
                                              cfg.pixel_size,
                                              cfg.grid_size_y_,
                                              cfg.grid_size_z_);

      cpu_image_sensitivity[px * cfg.n_pixels + py] =
          sensitivity(pixel_coordiantes.x,
                      pixel_coordiantes.y,
                      cfg.R_distance,
                      cfg.Scentilator_length / 2.0f);
    }
  }

  printf("CPU %f\n",
         sensitivity(100, 100, cfg.R_distance, cfg.Scentilator_length / 2.0f));

  float* gpu_image_buffor;
  float* gpu_image_rho;

  event<float>* gpu_event_list;

  // declare and allocate memory
  float* texture_sensitivity_buffer;

  size_t pitch;
  hipMallocPitch(&texture_sensitivity_buffer,
                  &pitch,
                  sizeof(float) * cfg.n_pixels,
                  cfg.n_pixels);

  printf("pitch = %d \n", pitch);
  hipMemcpy2D(texture_sensitivity_buffer,
               pitch,
               &cpu_image_sensitivity,
               sizeof(float) * cfg.n_pixels,
               sizeof(float) * cfg.n_pixels,
               cfg.n_pixels,
               hipMemcpyHostToDevice);

  //  cuda(Memcpy,
  //       texture_sensitivity_buffer,
  //       cpu_image_sensitivity,
  //       image_sz,
  //       hipMemcpyHostToDevice);

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = texture_sensitivity_buffer;
  resDesc.res.pitch2D.pitchInBytes = pitch;
  resDesc.res.pitch2D.width = cfg.n_pixels;
  resDesc.res.pitch2D.height = cfg.n_pixels;
  // resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
  resDesc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
  resDesc.res.pitch2D.desc.x = 32;  // 32 bits per channel for float texture
  resDesc.res.pitch2D.desc.y = 0;   // set 32 bits ONLY for float2 texture
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  //  hipResourceDesc resDesc;
  //  memset(&resDesc, 0, sizeof(resDesc));
  //  resDesc.resType = hipResourceTypeLinear;
  //  resDesc.res.linear.devPtr = texture_sensitivity_buffer;
  //  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  //  resDesc.res.linear.desc.x = 32;  // bits per channel
  //  resDesc.res.linear.sizeInBytes = image_sz;

  // other mallocs and allocations
  cuda(Malloc, (void**)&gpu_event_list, event_size * sizeof(event<float>));

  cuda(Malloc, (void**)&gpu_image_buffor, image_sz * cfg.number_of_blocks);

  cuda(Malloc, (void**)&gpu_image_rho, image_sz);

  cuda(Memcpy,
       gpu_event_list,
       event_list,
       event_size * sizeof(event<float>),
       hipMemcpyHostToDevice);

  cuda(Memcpy,
       gpu_image_buffor,
       cpu_image_buffor,
       image_sz * cfg.number_of_blocks,
       hipMemcpyHostToDevice);

  cuda(Memcpy, gpu_image_rho, cpu_image_rho, image_sz, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  reconstruction_2d_strip_cuda<float> << <blocks, threads>>>
      (cfg, gpu_event_list, event_size, gpu_image_buffor, gpu_image_rho, tex);

  hipDeviceSynchronize();

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Direct kernel time without memcpy %f ms\n", milliseconds);

  cuda(Memcpy,
       cpu_image_buffor,
       gpu_image_buffor,
       image_sz * cfg.number_of_blocks,
       hipMemcpyDeviceToHost);

  printf("HERE\n");

  for (int block_id = 0; block_id < cfg.number_of_blocks; ++block_id) {
    for (int index = 0; index < cfg.n_pixels * cfg.n_pixels; ++index) {

      image_output[index] +=
          cpu_image_buffor[block_id * cfg.n_pixels * cfg.n_pixels + index];

      if (image_output[index] > 0) {
        // printf("%d %d %f\n",1,1, image_output[index]);
      }
    }
  }

  cuda(DestroyTextureObject, tex);
  cuda(Free, gpu_image_buffor);
  cuda(Free, gpu_image_rho);
  cuda(Free, texture_sensitivity_buffer);
  // free(cpu_image_sensitivity);
  free(cpu_image_buffor);
  free(cpu_image_rho);
}
