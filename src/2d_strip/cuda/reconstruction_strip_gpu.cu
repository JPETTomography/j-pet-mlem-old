#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include "reconstruction.cuh"

static hipError_t err;

#define cuda_kernel_config(blocks, threads)                      \
  {                                                              \
    printf("Cuda kernel config\n");                              \
    printf("Number of  blocks per kernel: %d\n", blocks);        \
    printf("Number of threads|block per kernel: %d\n", threads); \
  }

#define cuda(f, ...)                                        \
  if ((err = cuda##f(__VA_ARGS__)) != hipSuccess) {        \
    fprintf(stderr, #f "() %s\n", hipGetErrorString(err)); \
    exit(-1);                                               \
  }
#define cudathread_per_blockoSync(...) cuda(__VA_ARGS__)

void gpu_reconstruction_strip_2d(gpu_config::GPU_parameters cfg,
                                 event<float>* event_list,
                                 int iteration_chunk) {

  hipSetDevice(0);

  dim3 blocks(cfg.number_of_blocks);
  dim3 threads(cfg.number_of_threads_per_block);

  cuda_kernel_config(cfg.number_of_blocks, cfg.number_of_threads_per_block);

  int event_list_size = 1000;

  int* cpu_example_data;

  cpu_example_data = (int*)malloc(1000 * sizeof(int));

  int* gpu_example_data;

  float* cpu_image_buffor =
      (float*)malloc(cfg.n_pixels * cfg.n_pixels * cfg.number_of_blocks);

  float* cpu_image_rho = (float*)malloc(cfg.n_pixels * cfg.n_pixels);

  float* gpu_image_buffor;
  float* gpu_image_rho;
  event<float>* gpu_event_list;

  cuda(Malloc, (void**)&gpu_example_data, 1000 * sizeof(int));

  cuda(Malloc,
       (void**)&gpu_image_buffor,
       cfg.n_pixels * cfg.n_pixels * cfg.number_of_blocks * sizeof(float));

  cuda(Malloc,
       (void**)&gpu_image_rho,
       cfg.n_pixels * cfg.n_pixels * sizeof(float));

  cuda(Malloc, (void**)&gpu_event_list, event_list_size * sizeof(event<float>));

  cuda(Memcpy,
       gpu_example_data,
       cpu_example_data,
       1000 * sizeof(int),
       hipMemcpyHostToDevice);

  cuda(Memcpy,
       gpu_image_buffor,
       cpu_image_buffor,
       cfg.n_pixels * cfg.n_pixels * cfg.number_of_blocks * sizeof(float),
       hipMemcpyHostToDevice);

  cuda(Memcpy,
       gpu_image_rho,
       cpu_image_rho,
       cfg.n_pixels * cfg.n_pixels * sizeof(float),
       hipMemcpyHostToDevice);

  cuda(Memcpy,
       gpu_event_list,
       event_list,
       event_list_size * sizeof(event<float>),
       hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  reconstruction_2d_strip_cuda<float> << <blocks, threads>>>
      (cfg, event_list, event_list_size, gpu_image_buffor, gpu_image_rho);

  hipDeviceSynchronize();

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Direct kernel time without memcpy %f ms\n", milliseconds);

  cuda(Free, gpu_example_data);
  cuda(Free, gpu_image_buffor);
  cuda(Free, gpu_image_rho);
  free(cpu_example_data);
  free(cpu_image_buffor);
  free(cpu_image_rho);
}
